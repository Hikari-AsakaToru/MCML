#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define _NVCC_
#include "CUDAMCML_GPGPU.h"
#include <hiprand/hiprand_kernel.h>

#define _ERR_GPU_SIM_RND_ 1
#define _ERR_GPU_SIM_MEMCPY_ 2
#define _ERR_GPU_SIM_LOOP_ 3
#define _ERR_GPU_SIM_LANCH_PHOTON_ 4
#define _ERR_GPU_SIM_MCML_ 5
#define _ERR_GPU_SIM_ANOTHER_ 0xFF
#define _SUCCESS_GPU_SIM_ 0


// MemStruct m_sDeviceMem;

__device__ __constant__ unsigned int num_photons_dc[1];
__device__ __constant__ unsigned int n_layers_dc[1];
__device__ __constant__ unsigned long long start_weight_dc[1];
__device__ __constant__ LayerStruct layers_dc[MAX_LAYERS];
__device__ __constant__ DetStruct det_dc[1];
__device__ __constant__ unsigned int dc_Seed[1];
__device__ unsigned int nInitRngLoop=0;

__shared__ PhotonStruct dsh_sPhoton[NUM_THREADS_PER_BLOCK];

//
// MCML�v�Z�̖{��
// 
template <int ignoreAdetection> __global__ void MCd(MemStruct DeviceMem)
{
	//Block index
	int bx = blockIdx.x;

	//Thread index
	int tx = threadIdx.x;


	//First element processed by the block
	int begin = blockDim.x*bx;



	unsigned long long int x = DeviceMem.x[begin + tx];//coherent
	unsigned int a = DeviceMem.a[begin + tx];//coherent

	float s;	//step length

	unsigned long long index, w, index_old,DataPos;
	index_old = 0;
	w = 0;
	unsigned int w_temp;
	DataPos = *DeviceMem.num_terminated_photons;

	PhotonStruct p = DeviceMem.p[begin + tx];
	

	int new_layer;

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;
	if (!DeviceMem.thread_active[begin + tx]){
		ii = NUMSTEPS_GPU;
	}

	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
	{
		if (layers_dc[p.layer].mutr != FLT_MAX)
			s = -__logf(rand_MWC_oc(&x, &a))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		else
			s = 100.0f;//temporary, say the step in glass is 100 cm.

		//Check for layer transitions and in case, calculate s
		new_layer = p.layer;
		if (p.z + s*p.dz<layers_dc[p.layer].z_min){ new_layer--; s = __fdividef(layers_dc[p.layer].z_min - p.z, p.dz); } //Check for upwards reflection/transmission & calculate new s
		if (p.z + s*p.dz>layers_dc[p.layer].z_max){ new_layer++; s = __fdividef(layers_dc[p.layer].z_max - p.z, p.dz); } //Check for downward reflection/transmission

		p.x += p.dx*s;
		p.y += p.dy*s;
		p.z += p.dz*s;

		if (p.z>layers_dc[p.layer].z_max)p.z = layers_dc[p.layer].z_max;//needed?
		if (p.z<layers_dc[p.layer].z_min)p.z = layers_dc[p.layer].z_min;//needed?

		if (new_layer != p.layer)
		{
			// set the remaining step length to 0
			s = 0.0f;

			if (Reflect(&p, new_layer, &x, &a) == 0u)//Check for reflection
			{ // Photon is transmitted
				if (new_layer == 0)
				{ //Diffuse reflectance
					index = __float2int_rz(acosf(-p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Rd_ra[index], p.weight);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
				if (new_layer > *n_layers_dc)
				{	//Transmitted
					index = __float2int_rz(acosf(p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Tt_ra[index], p.weight);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
			}
		}

		//w=0;

		if (s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered)
			w_temp = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			p.weight -= w_temp;


			//w = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			//p.weight -= w;

			if (ignoreAdetection == 0) // Evaluated at compiletime!
			{
				index = (min(__float2int_rz(__fdividef(p.z, det_dc[0].dz)), (int)det_dc[0].nz - 1)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1));
				if (index == index_old)
				{
					w += w_temp;
					//p.weight -= __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight)); 
				}
				else// if(w!=0)
				{
					AtomicAddULL(&DeviceMem.A_rz[index_old], w);
					index_old = index;
					w = w_temp;
				}

			}

			Spin(&p, &x, &a, layers_dc[p.layer].g);
		}




		if (!PhotonSurvive(&p, &x, &a)) // Check if photons survives or not
		{
			if (atomicAdd(DeviceMem.num_terminated_photons, 1u) < (*num_photons_dc))
			{	// Ok to launch another photon
				LaunchPhoton(&p);//Launch a new photon
			}
			else
			{	// No more photons should be launched. 
				DeviceMem.thread_active[DataPos] = 0u; // Set thread to inactive
				ii = NUMSTEPS_GPU;				// Exit main loop
			}

		}
	}//end main for loop!
	if (ignoreAdetection == 1 && w != 0)
		AtomicAddULL(&DeviceMem.A_rz[index_old], w);

	__syncthreads();//necessary?

	//save the state of the MC simulation in global memory before exiting
	DeviceMem.p[DataPos] = p;	//This one is incoherent!!!
	DeviceMem.x[DataPos] = x; //this one also seems to be coherent


}//end MCd
template <int ignoreAdetection> __global__ void CalcMCGPU(MemStruct DeviceMem)
{
	//Block index
	int bx = blockIdx.x;

	//Thread index
	int tx = threadIdx.x;


	//First element processed by the block
	int begin = blockDim.x*bx;
	if (DeviceMem.thread_active[begin + tx] == 65535){
		return;
	}
	if (DeviceMem.thread_active[begin + tx]){
		auto temp = atomicAdd(DeviceMem.num_terminated_photons, 1ul);
		if (temp > *num_photons_dc){
			DeviceMem.thread_active[begin + tx] = 65535;
			return;
		}
		DeviceMem.thread_active[begin + tx] = 0;
	}

	DeviceMem.thread_active[begin + tx] = 0;
	unsigned long long int x = DeviceMem.x[begin + tx];	//coherent
	unsigned int a = DeviceMem.a[begin + tx];			//coherent
	dsh_sPhoton[tx] = DeviceMem.p[begin + tx];
	float s;											//step length

	unsigned int index, index_old;
	index_old = 0;
	unsigned long long w,w_temp;
	w = 0;

	int new_layer;

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;


	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
	{
		// Rand Make
		// �����ӂ�o�Ȃ����m�F
		if (layers_dc[dsh_sPhoton[tx].layer].mutr != FLT_MAX){
			// �����ɂ�鋗������
			s = -__logf(rand_MWC_oc(&x, &a))*layers_dc[dsh_sPhoton[tx].layer].mutr;	//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		}else{
			// �ꎞ�I��100 cm���
			s = 100.0f;															//temporary, say the step in glass is 100 cm.
		}
		// Hop_Drop() mcml_go
		//Check for layer transitions and in case, calculate s
		new_layer = dsh_sPhoton[tx].layer;
		// ���݂̃��C���[������Ɉړ����Ă邩���m�F
		if (dsh_sPhoton[tx].z + s*dsh_sPhoton[tx].dz<layers_dc[dsh_sPhoton[tx].layer].z_min){
			new_layer--; 
			s = __fdividef(layers_dc[dsh_sPhoton[tx].layer].z_min - dsh_sPhoton[tx].z, dsh_sPhoton[tx].dz); 
		} //Check for upwards reflection/transmission & calculate new s
		// ���݂̃��C���[�������Ɉړ����Ă邩���m�F
		if (dsh_sPhoton[tx].z + s*dsh_sPhoton[tx].dz>layers_dc[dsh_sPhoton[tx].layer].z_max){
			new_layer++;
			s = __fdividef(layers_dc[dsh_sPhoton[tx].layer].z_max - dsh_sPhoton[tx].z, dsh_sPhoton[tx].dz); 
		} //Check for downward reflection/transmission

		// �ʒu����
		dsh_sPhoton[tx].x += dsh_sPhoton[tx].dx*s;
		dsh_sPhoton[tx].y += dsh_sPhoton[tx].dy*s;
		dsh_sPhoton[tx].z += dsh_sPhoton[tx].dz*s;
//		Hop(&dsh_sPhoton[tx],s);
		if (dsh_sPhoton[tx].z > layers_dc[dsh_sPhoton[tx].layer].z_max){
			dsh_sPhoton[tx].z = layers_dc[dsh_sPhoton[tx].layer].z_max;//needed?
		}
		if (dsh_sPhoton[tx].z < layers_dc[dsh_sPhoton[tx].layer].z_min){
			dsh_sPhoton[tx].z = layers_dc[dsh_sPhoton[tx].layer].z_min;//needed?
		}
		//�@���C���[�ω����Ă����ꍇ

		if (new_layer != dsh_sPhoton[tx].layer)
		{
			// set the remaining step length to 0
			s = 0.0f;

			// ���˂��邩�m�F
			if (Reflect(&dsh_sPhoton[tx], new_layer, &x, &a) == 0u)//Check for reflection
			{ 
				// Photon is transmitted�@���q���`�B
				if (new_layer == 0)
				{	// Diffuse reflectance�@�g�U����
					// __float2int_rz �E�E�Efloat  => int�@�ւ̌^�ϊ�(�����_�؂�̂āH)
//					index = __float2int_rz(acosf(-dsh_sPhoton[tx].dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					index = __float2int_rz(__fdividef(acosf(-dsh_sPhoton[tx].dz) , (PI / det_dc[0].na)))*det_dc[0].nr + min(__float2int_rz(__fdividef(__fsqrt_rz(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Rd_ra[index], dsh_sPhoton[tx].weight);
					dsh_sPhoton[tx].weight = 0;
				}
				if (new_layer > *n_layers_dc)
				{	//Transmitted�@����
					index = __float2int_rz(__fdividef(acosf(dsh_sPhoton[tx].dz), (PI / det_dc[0].na)))*det_dc[0].nr + min(__float2int_rz(__fdividef(__fsqrt_rz(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Tt_ra[index], dsh_sPhoton[tx].weight);
					dsh_sPhoton[tx].weight = 0;
				}
			}
		}
		//w=0;
		if (s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered) ���q�̎��ʌ���
			w_temp = __float2uint_rn(layers_dc[dsh_sPhoton[tx].layer].mua*layers_dc[dsh_sPhoton[tx].layer].mutr*__uint2float_rn(dsh_sPhoton[tx].weight));
			dsh_sPhoton[tx].weight -= w_temp;

			if (ignoreAdetection == 0) // Evaluated at compiletime!
			{
				index = (min(__float2int_rz(__fdividef(dsh_sPhoton[tx].z, det_dc[0].dz)), (int)det_dc[0].nz - 1)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1));
				if (index == index_old)
				{
					w += w_temp;
					//sharedp.weight -= __float2uint_rn(layers_dc[sharedp.layer].mua*layers_dc[sharedp.layer].mutr*__uint2float_rn(sharedp.weight)); 
				}
				else// if(w!=0)
				{
					AtomicAddULL(&DeviceMem.A_rz[index_old], w);
					index_old = index;
					w = w_temp;
				}

			}
			// �p�x�v�Z
			Spin(&dsh_sPhoton[tx], &x, &a, layers_dc[dsh_sPhoton[tx].layer].g);
		}



		if (!PhotonSurvive(&dsh_sPhoton[tx], &x, &a)) // Check if photons survives or not
		{
			DeviceMem.thread_active[begin + tx] = 1;
			LaunchPhoton(&dsh_sPhoton[tx]);
			break;
		}
	}//end main for loop!
	if (ignoreAdetection == 1 && w != 0)
		AtomicAddULL(&DeviceMem.A_rz[index_old], w);

	__syncthreads();//necessary?

	DeviceMem.x[begin + tx] = x; //this one also seems to be coherent
	DeviceMem.p[begin + tx] = dsh_sPhoton[tx]; //this one also seems to be coherent


}//end MCd

__device__  void LaunchPhoton(PhotonStruct* p)
{
	// We are currently not using the RNG but might do later
	//float input_fibre_radius = 0.03;//[cm]
	//p->x=input_fibre_radius*sqrtf(rand_MWC_co(x,a));

	p->x = 0.0f;
	p->y = 0.0f;
	p->z = 0.0f;
	p->dx = 0.0f;
	p->dy = 0.0f;
	p->dz = 1.0f;

	p->layer = 1;
	p->weight = *start_weight_dc; //specular reflection!

}

__global__ void LaunchPhoton_Global(PhotonStruct* pd)
{
	unsigned long long PosData = blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x;
	//First element processed by the block
	if (PosData < num_photons_dc[0]){
	
		//	 LaunchPhoton(&pd[PosData], d_x[PosData], d_a[PosData]);
		pd[PosData].dx = 0.0f;
		pd[PosData].dy = 0.0f;
		pd[PosData].dz = 1.0f;
		pd[PosData].x	= 0.0f;
		pd[PosData].y	= 0.0f;
		pd[PosData].z	= 0.0;
		pd[PosData].layer	= 1;
		pd[PosData].Index = PosData;
		pd[PosData].weight	= (unsigned int)*start_weight_dc;

		//DeviceMem->p[begin + tx] = p;//incoherent!?
		
	}
	return;
}
__global__ void SetRandpram(hiprandState* devState){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/* Each thread gets same seed, a different sequence number,
	no offset */
	hiprand_init(*dc_Seed, id, 0, &devState[id]);
}

__global__ void InitRng(MemStruct devMem,hiprandState* RndMakerglobal){
	hiprandState RndMakerLocal;
	RndMakerLocal = RndMakerglobal[threadIdx.x];
	unsigned long long* X = devMem.x;
	unsigned int* A = devMem.a;
	unsigned long long un64PosData = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int TmpRndH, TmpRndL;
	TmpRndH = hiprand(&RndMakerLocal);
	__syncthreads();
	TmpRndL = hiprand(&RndMakerLocal);
	unsigned long long TmpRAXH = ((unsigned long long)TmpRndH) << 32;
	X[un64PosData] = TmpRAXH | (unsigned long long)TmpRndL;
	__syncthreads();
	TmpRndL = hiprand(&RndMakerLocal);
	A[un64PosData] = TmpRndL;
	un64PosData += NUM_DIV_MAKE_RAND*NUM_THREADS_PER_BLOCK_MAKE_RAND;


		

	return;
}
__device__ double SpinTheta(unsigned long long int* x, unsigned int *a, double g){
	double cost;

	if (g == 0.0)
		cost = 2 * rand_MWC_co(x, a) - 1;
	else {
		double temp = (1 - g*g) / (1 - g + 2 * g*rand_MWC_co(x, a));
		cost = (1 + g*g - temp*temp) / (2 * g);
		if (cost < -1) cost = -1;
		else if (cost > 1) cost = 1;
	}
	return(cost);
}
__device__ void Hop(PhotonStruct* p,float s){
	p->x = s*p->dx;
	p->y = s*p->dy;
	p->z = s*p->dz;
}
__device__ void Spin(PhotonStruct* p, unsigned long long int* x, unsigned int *a, float g)
{
	float cost, sint;	// cosine and sine of the 
	// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
	// azimuthal angle psi. 
	float temp=2.1;

	float tempdir = p->dx;
	// Open CUDA Code
	//This is more efficient for g!=0 but of course less efficient for g==0
	//	temp = __fdividef((1.0f - (g)*(g)), (1.0f - (g)+2.0f*(g)*rand_MWC_co(x, a)));//Should be close close????!!!!!
	//	cost = __fdividef((1.0f + (g)*(g)-temp*temp), (2.0f*(g)));
	//	if (g == 0.0f)
	//		cost = 2.0f*rand_MWC_co(x, a) - 1.0f;//Should be close close??!!!!!

	// MIYAHIRA mcml SpinTheta()
	cost = SpinTheta(x,a,g);

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rand_MWC_co(x, a), &sinp, &cosp);// spin psi [0-2*PI)

	temp = sqrtf(1.0f - p->dz*p->dz);

	if (temp == 0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost, p->dz*cost);	// copysign(a,b)==  a*SIGN(b) 
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp), temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp), temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp = rsqrtf(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;

}// end Spin
__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, unsigned long long* x, unsigned int* a)
{
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)


	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);

	if (n1 == n2)//refraction index matching automatic transmission and no direction change
	{
		p->layer = new_layer;
		return 0u;
	}

	if (n1>n2 && n2*n2<n1*n1*(1 - cos_angle_i*cos_angle_i))//total internal reflection, no layer change but z-direction mirroring
	{
		p->dz *= -1.0f;
		return 1u;
	}

	if (cos_angle_i == 1.0f)//normal incident
	{
		r = __fdividef((n1 - n2), (n1 + n2));
		if (rand_MWC_co(x, a) <= r*r)
		{
			//reflection, no layer change but z-direction mirroring
			p->dz *= -1.0f;
			return 1u;
		}
		else
		{	//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
		}
	}
	else
	{
		//long and boring calculations of r
		float sinangle_i = sqrtf(1.0f - p->dz*p->dz);
		float sinangle_e = n1/n2*sinangle_i;
		float cosangle_e = sqrtf(1.0f - sinangle_e*sinangle_e);

		float cossumangle = (p->dz*cosangle_e) - sinangle_i*sinangle_e;
		float cosdiffangle = (p->dz*cosangle_e) + sinangle_i*sinangle_e;
		float sinsumangle = sinangle_i*cosangle_e + (p->z*sinangle_e);
		float sindiffangle = sinangle_i*cosangle_e - (p->z*sinangle_e);

		r = 0.5*sindiffangle*sindiffangle*__fdividef((cosdiffangle*cosdiffangle + cossumangle*cossumangle), (sinsumangle*sinsumangle*cosdiffangle*cosdiffangle));

	}
	//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
	// save a few multiplications, calculate cos_angle_i^2;
	//float e = __fdividef(n1*n1, n2*n2)*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
	//r = 2 * sqrtf((1.0f - cos_angle_i*cos_angle_i)*(1.0f - e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
	//e = e + (cos_angle_i*cos_angle_i)*(1.0f - 2.0f*e);//Update the value of e
	//r = e*__fdividef((1.0f - e - r), ((1.0f - e + r)*(e + r)));//Calculate r	

	if (rand_MWC_co(x, a) <= r)
	{
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		return 1u;
	}
	else
	{
		// Transmission, update layer and direction
		r = __fdividef(n1, n2);
		float e = r*r*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1 - e), p->dz);
		p->layer = new_layer;
		return 0u;
	}

}
__device__ unsigned int PhotonSurvive(PhotonStruct* p, unsigned long long* x, unsigned int* a)
{	//Calculate wether the photon survives (returns 1) or dies (returns 0)

	if (p->weight>WEIGHTI) return 1u; // No roulette needed
	if (p->weight == 0u) return 0u;	// Photon has exited slab, i.e. kill the photon

	if (rand_MWC_co(x, a)<CHANCE)
	{
		p->weight = __float2uint_rn(__fdividef((float)p->weight, CHANCE));
		return 1u;
	}

	//else
	return 0u;
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned long long add)
{
	if (atomicAdd((unsigned long long*)address, add) + add<add)
		atomicAdd(((unsigned long long*)address) + 1, 1u);
}
__device__ float rand_MWC_co(unsigned long long* x, unsigned int* a)
{
	float temp = 0.0;
	//Generate a random number [0,1)
	*x = (*x & 0xffffffffull)*(*a) + (*x >> 32);
	temp = __fdividef(__uint2float_rz((unsigned int)(*x)), (float)0x100000000);// The typecast will truncate the x so that it is 0<=x<(2^32-1),__uint2float_rz ensures a round towards zero since 32-bit floating point cannot represent all integers that large. Dividing by 2^32 will hence yield [0,1)
	return temp;
}//end __device__ rand_MWC_co
__device__ float rand_MWC_oc(unsigned long long* x, unsigned int* a)
{
	//Generate a random number (0,1]
	return 1.0f - rand_MWC_co(x, a);
}//end __device__ rand_MWC_oc


cCUDAMCML::cCUDAMCML(){

}
cCUDAMCML::~cCUDAMCML(){
}
void cCUDAMCML::RunOldCarnel(){
	dim3 dimGrid(NUM_GRID_PER_BLOCK);
	dim3 dimBlock(NUM_THREADS_PER_BLOCK);
	unsigned int threads_active_total = 1;
	int i = 0;
	while (*m_sHostMem.num_terminated_photons < m_simulations->number_of_photons)
	{
		i++;
		//run the kernel
		if (m_simulations->ignoreAdetection == 1){
			MCd<1> << <dimGrid, dimBlock >> >(m_sDeviceMem);
		}
		else{
			MCd<0> << <dimGrid, dimBlock >> >(m_sDeviceMem);
		}
		hipDeviceSynchronize(); // Wait for all threads to finish
		hipError_t cudastat = hipGetLastError(); // Check if there was an error

		// Copy thread_active from device to host
		hipMemcpy(m_sHostMem.thread_active, m_sDeviceMem.thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyDeviceToHost);
		threads_active_total = 0;
		for (int ii = 0; ii < NUM_THREADS; ii++){
			threads_active_total += m_sHostMem.thread_active[ii];
		}

		hipMemcpy(m_sHostMem.num_terminated_photons, m_sDeviceMem.num_terminated_photons, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(m_sHostMem.p, m_sDeviceMem.p, sizeof(PhotonStruct)*m_simulations->number_of_photons, hipMemcpyDeviceToHost);
	}
	CopyDeviceToHostMem(&m_sHostMem, &m_sDeviceMem, m_simulations);
}
// �v�Z�̒���
int cCUDAMCML::MakeRandTableDev(){
	hiprandState *devStates;
	hipError_t  cudastat;
	dim3 dimNumBlockRand(NUM_GRID_PER_BLOCK);
	dim3 dimNumThreadRand(NUM_THREADS_PER_BLOCK);
	hipMalloc((void **)&devStates, NUM_THREADS * sizeof(hiprandState));
	// �V�[�h�C�����l�Ƃ��ėp���闐���z��̍쐬
	// MCML�̗��������ɗ��p�ł��Ȃ��@�ˁ@���C�u�����̓s����C�X���b�h������������邽��
	SetRandpram << < dimNumBlockRand, dimNumThreadRand >> > (devStates);
	cudastat = hipGetLastError();	// Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_RND_;
	}
	hipDeviceSynchronize();
	InitRng << < dimNumBlockRand, dimNumThreadRand >> > (m_sDeviceMem,devStates);
	// ���ؗp
	hipMemcpy(m_sHostMem.a, m_sDeviceMem.a, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(m_sHostMem.x, m_sDeviceMem.x, NUM_THREADS * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	cudastat = hipGetLastError();	// Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_RND_;
	}
	return 0;
}
int cCUDAMCML::InitPhoton(){


	dim3 dimNumBlock(19);
	dim3 dimNumThread(NUM_THREADS_PER_BLOCK);

	LaunchPhoton_Global << < dimNumBlock, dimNumThread >> > (m_sDeviceMem.p);
	hipError_t cudastat = hipGetLastError();	// Check if there was an error

	if (cudastat){
		return _ERR_GPU_SIM_LANCH_PHOTON_;
	}
	return 0;
}
int cCUDAMCML::DoOneSimulation(SimulationStruct* simulation)
{

	unsigned int threads_active_total = simulation->number_of_photons;
	unsigned int i;

	hipError_t cudastat;

	int STAT = 0;
	// Start the clock

	dim3 dimNumBlock(NUM_GRID_PER_BLOCK);
	dim3 dimNumThread(NUM_THREADS_PER_BLOCK);
	int TotalP = 0;
	while (TotalP<simulation->number_of_photons){
		//run the kernel
		if (simulation->ignoreAdetection == 1){
			CalcMCGPU<1> << < dimNumBlock, dimNumThread >> >(m_sDeviceMem);
	
		}
		else{
			CalcMCGPU<0> << < dimNumBlock, dimNumThread >> >(m_sDeviceMem);
	
		}
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat){
			return _ERR_GPU_SIM_MCML_;
		}
		// ���ؗp
		cudastat = hipMemcpy(m_sHostMem.p, m_sDeviceMem.p, NUM_THREADS * sizeof(PhotonStruct), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.thread_active, m_sDeviceMem.thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.num_terminated_photons, m_sDeviceMem.num_terminated_photons,sizeof(unsigned int), hipMemcpyDeviceToHost);
		//	hipDeviceSynchronize();		// Wait for all threads to finish
		// 
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat){
			return _ERR_GPU_SIM_MEMCPY_;
		}
		for (int i = 0; i < NUM_THREADS; i++){
			if (m_sHostMem.thread_active[i] != 65535){
				TotalP += m_sHostMem.thread_active[i];
			}
		}
	}

	cudastat = hipGetLastError(); // Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_ANOTHER_;
	}


	CopyDeviceToHostMem(&m_sHostMem, &m_sDeviceMem, simulation);
	return _SUCCESS_GPU_SIM_;

}
int cCUDAMCML::InitMallocMem(SimulationStruct* sim){
	unsigned char State = 0;
	hipError_t tmp;

	tmp = hipMalloc((void**)&m_sDeviceMem, sizeof(MemStruct));
	if (tmp != hipSuccess) {
		State |= 0x01;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.p, (NUM_THREADS)*sizeof(PhotonStruct));
	if (tmp != hipSuccess) {
		State |= 0x01;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.x, (NUM_THREADS)*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x02;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.a, (NUM_THREADS)*sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x04;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.thread_active, (NUM_THREADS)*sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x08;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.num_terminated_photons, sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x10;
	}
	int rz_size = sim->det.nr*sim->det.nz;
	int ra_size = sim->det.nr*sim->det.na;
	tmp = hipMalloc((void**)&m_sDeviceMem.A_rz, rz_size *sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.Rd_ra, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x40;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.Tt_ra, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x80;
	}


	// Allocate p on the device!!
	// Allocate A_rz on host and device
	m_sHostMem.p = new PhotonStruct			[NUM_THREADS];
	m_sHostMem.x = new unsigned long long	[NUM_THREADS];
	m_sHostMem.a = new unsigned int			[NUM_THREADS];
	if ((m_sHostMem.x != NULL) && (m_sHostMem.a != NULL)){
		State |= 0x00200000;
	}
	m_sHostMem.A_rz = new unsigned long long [rz_size];
	if (m_sHostMem.A_rz == NULL){
		State |= 0x00010000;
	}
	m_sHostMem.Rd_ra = new unsigned long long[ra_size];
	if (m_sHostMem.Rd_ra == NULL){
		State |= 0x00020000;
	}
	m_sHostMem.Tt_ra = new unsigned long long[ra_size];
	if (m_sHostMem.Tt_ra == NULL){
		State |= 0x00040000;
	}
	// Allocate thread_active on the device and host
	m_sHostMem.thread_active = new unsigned int[sim->number_of_photons];
	if (m_sHostMem.thread_active == NULL){
		State |= 0x00080000;
	}

	m_sHostMem.num_terminated_photons = new unsigned int[1];
	if (m_sHostMem.num_terminated_photons == NULL){
		State |= 0x00100000;
	}
	*m_sHostMem.num_terminated_photons = 0;




	return State;
}
void cCUDAMCML::CopyDeviceToHostMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{ //Copy data from Device to Host memory

	int rz_size = sim->det.nr*sim->det.nz;
	int ra_size = sim->det.nr*sim->det.na;

	// Copy A_rz, Rd_ra and Tt_ra
	hipMemcpy(HostMem->A_rz, DeviceMem->A_rz, rz_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);
	hipMemcpy(HostMem->Rd_ra, DeviceMem->Rd_ra, ra_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);
	hipMemcpy(HostMem->Tt_ra, DeviceMem->Tt_ra, ra_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);

	//Also copy the state of the RNG's
	hipMemcpy(HostMem->p, DeviceMem->p, NUM_THREADS *sizeof(PhotonStruct), hipMemcpyDeviceToHost);

	return ;
}
int cCUDAMCML::CopyHostToDeviceMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim){
	// Allocate x and a on the device (For MWC RNG)
	hipError_t tmp;
	int State = 0;
	tmp = hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS*sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x40;
	}


	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x80;
	}
	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x200;
	}
	tmp = hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x800;
	}
	return State;
}
int cCUDAMCML::InitDCMem(SimulationStruct* sim)
{
	hipError_t tmp;
	int State = 0;
	// Copy det-data to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(det_dc), &(sim->det), sizeof(DetStruct));
	if (tmp != hipSuccess) {
		State |= 0x1;
	}

	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc), &(sim->n_layers), sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x2;
	}

	// Copy start_weight_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc), &(sim->start_weight), sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x4;
	}

	// Copy layer data to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(layers_dc), sim->layers, (sim->n_layers + 2)*sizeof(LayerStruct));
	if (tmp != hipSuccess) {
		State |= 0x8;
	}

	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc), &(sim->number_of_photons), sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x10;
	}
	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(dc_Seed), &(sim->Seed), sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}

	return State;

}
int cCUDAMCML::InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{
	int rz_size, ra_size;
	hipError_t tmp;
	rz_size = sim->det.nr*sim->det.nz;
	ra_size = sim->det.nr*sim->det.na;
	int Total = 0;

	// Allocate p on the device!!
	hipMalloc((void**)&DeviceMem->p, NUM_THREADS*sizeof(PhotonStruct));
	Total += NUM_THREADS*sizeof(PhotonStruct);
	// Allocate A_rz on host and device
	HostMem->A_rz = new(unsigned long long)(rz_size*sizeof(unsigned long long));
	if (HostMem->A_rz == NULL){ 
		printf("Error allocating HostMem->A_rz"); 
		exit(1); 
	}
	hipMalloc((void**)&DeviceMem->A_rz, rz_size*sizeof(unsigned long long));
	Total += rz_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->A_rz, 0, rz_size*sizeof(unsigned long long));

	// Allocate Rd_ra on host and device
	HostMem->Rd_ra = new(unsigned long long)(ra_size*sizeof(unsigned long long));
	if (HostMem->Rd_ra == NULL){ 
		printf("Error allocating HostMem->Rd_ra"); 
		exit(1);
	}
	hipMalloc((void**)&DeviceMem->Rd_ra, ra_size*sizeof(unsigned long long));
	Total += ra_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->Rd_ra, 0, ra_size*sizeof(unsigned long long));

	// Allocate Tt_ra on host and device
	HostMem->Tt_ra = new(unsigned long long)(ra_size*sizeof(unsigned long long));
	if (HostMem->Tt_ra == NULL){ printf("Error allocating HostMem->Tt_ra"); exit(1); }
	hipMalloc((void**)&DeviceMem->Tt_ra, ra_size*sizeof(unsigned long long));
	Total += ra_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->Tt_ra, 0, ra_size*sizeof(unsigned long long));


	// Allocate x and a on the device (For MWC RNG)
	hipMalloc((void**)&DeviceMem->x, NUM_THREADS*sizeof(unsigned long long));
	Total += NUM_THREADS*sizeof(unsigned long long);

	hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS*sizeof(unsigned long long), hipMemcpyHostToDevice);
	tmp = hipMalloc((void**)&DeviceMem->a, NUM_THREADS*sizeof(unsigned int));
	Total += NUM_THREADS*sizeof(unsigned long long);

	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);


	// Allocate thread_active on the device and host
	HostMem->thread_active = new unsigned int (NUM_THREADS*sizeof(unsigned int));
	if (HostMem->thread_active == NULL){ printf("Error allocating HostMem->thread_active"); exit(1); }
	for (int i = 0; i < NUM_THREADS; i++){
		HostMem->thread_active[i] = 1u;
	}

	Total += NUM_THREADS*sizeof(unsigned int);
	tmp = hipMalloc((void**)&(DeviceMem->thread_active), NUM_THREADS*sizeof(unsigned int));

	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);


	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = new(unsigned int)(sizeof(unsigned int));
	if (HostMem->num_terminated_photons == NULL){ printf("Error allocating HostMem->num_terminated_photons"); exit(1); }
	*HostMem->num_terminated_photons = 0;

	hipMalloc((void**)&DeviceMem->num_terminated_photons, sizeof(unsigned int));
	hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);

	return 1;
}
int cCUDAMCML::InitContentsMem(SimulationStruct* sim)
{
	MemStruct* DeviceMem = &m_sDeviceMem;
	MemStruct* HostMem = &m_sHostMem;

	int State=0;
	int rz_size, ra_size;
	hipError_t tmp;
	rz_size = sim->det.nr*sim->det.nz;
	ra_size = sim->det.nr*sim->det.na;

	tmp = hipMemset(DeviceMem->A_rz, 0, rz_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x02;
	}

	tmp = hipMemset(DeviceMem->Rd_ra, 0, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x08;
	}

	tmp = hipMemset(DeviceMem->Tt_ra, 0, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}

	PhotonStruct TmpPS;

	HostMem->p->x = 0;
	HostMem->p->y = 0;
	HostMem->p->z = 0;
	HostMem->p->dx = 0;
	HostMem->p->dy = 0;
	HostMem->p->dz = 0;
	HostMem->p->weight = 0;
	HostMem->p->layer = 0;

	tmp = hipMemset(DeviceMem->p, 0, NUM_THREADS *sizeof(PhotonStruct));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}

	tmp = hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS *sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x40;
	}


	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x80;
	}

	for (int i = 0; i < sim->number_of_photons; i++){
		HostMem->thread_active[i] = 1u;
	}



	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x200;
	}



	tmp = hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x800;
	}

	return State;
}

void cCUDAMCML::FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{

	hipFree(DeviceMem->p);
	hipFree(DeviceMem->x);
	hipFree(DeviceMem->a);
	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);
	hipFree(DeviceMem->A_rz);
	hipFree(DeviceMem->Rd_ra);
	hipFree(DeviceMem->Tt_ra);
	hipFree(DeviceMem);
	delete[] HostMem->p;
	delete[] HostMem->x;
	delete[] HostMem->a;
	delete[] HostMem->thread_active;
	delete[] HostMem->num_terminated_photons;
	delete[] HostMem->Reserve;
	delete[] HostMem->A_rz;
	delete[] HostMem->Rd_ra;
	delete[] HostMem->Tt_ra;


}

void cCUDAMCML::FreeSimulationStruct(SimulationStruct* sim, int nRun)
{
	FreeMemStructs(&m_sHostMem, &m_sDeviceMem);
	for (int i = 0; i < nRun; i++){
		delete sim[i].layers;
	}
	delete[] sim;
	//hipDeviceReset();
}

void cCUDAMCML::FreeFailedSimStrct(SimulationStruct* Sim, int nRun)
{
	for (int i = 0; i < nRun; i++){
		delete Sim[i].layers;
	}
	delete[] Sim;

}


bool cCUDAMCML::CheckGPU(){
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount==0){
		return 0;
	}
	hipSetDevice(0);
	hipGetDeviceProperties(&m_sDevProp, 0);
	return 1;
}
void cCUDAMCML::InitGPUStat(){
	m_ProcessTime = 0;
	m_un64Membyte = 0;
	m_un64NumPhoton = 0;
	m_un64PrcsDataNum = 0;
	hipDeviceReset();
}
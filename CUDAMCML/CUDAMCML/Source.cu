#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define _NVCC_
#include "CUDAMCML_GPGPU.h"
#include <hiprand/hiprand_kernel.h>

#define _ERR_GPU_SIM_RND_ 1
#define _ERR_GPU_SIM_MEMCPY_ 2
#define _ERR_GPU_SIM_LOOP_ 3
#define _ERR_GPU_SIM_LANCH_PHOTON_ 4
#define _ERR_GPU_SIM_MCML_ 5
#define _ERR_GPU_SIM_ANOTHER_ 0xFF
#define _SUCCESS_GPU_SIM_ 0
#define PARTIALREFLECTION 1     
#define GNUCC 0
/* 1=split photon, 0=statistical reflection. */

// MemStruct m_sDeviceMem;

__device__ __constant__ unsigned int num_photons_dc[1];
__device__ __constant__ unsigned int n_layers_dc[1];
__device__ __constant__ unsigned long long start_weight_dc[1];
__device__ __constant__ LayerStruct layers_dc[MAX_LAYERS];
__device__ __constant__ DetStruct det_dc[1];
__device__ __constant__ unsigned int dc_Seed[1];
__device__ unsigned int nInitRngLoop=0;

__shared__ PhotonStruct dsh_sPhoton[NUM_THREADS_PER_BLOCK];


//
// MCML�v�Z�̖{��
// 
template <int ignoreAdetection> __global__ void MCd(MemStruct DeviceMem)
{
	//Block index
	int bx = blockIdx.x;

	//Thread index
	int tx = threadIdx.x;


	//First element processed by the block
	int begin = blockDim.x*bx;



	unsigned long long int x = DeviceMem.x[begin + tx];//coherent
	unsigned int a = DeviceMem.a[begin + tx];//coherent

	float s;	//step length
	//p->s = s;   //PhotonStruct��s�Ƃ̓������K�v�H

	unsigned long long index, w, index_old,DataPos;
	index_old = 0;
	w = 0;
	unsigned int w_temp;
	DataPos = *DeviceMem.num_terminated_photons;

	PhotonStruct p = DeviceMem.p[begin + tx];
	

	int new_layer;

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;
	if (!DeviceMem.thread_active[begin + tx]){
		ii = NUMSTEPS_GPU;
	}

	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
	{
		if (layers_dc[p.layer].mutr != FLT_MAX)
			p.s = -__logf(rand_MWC_oc(&x, &a))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		else
			p.s = 100.0f;//temporary, say the step in glass is 100 cm.

		//Check for layer transitions and in case, calculate s
		new_layer = p.layer;
		if (p.z + s*p.dz<layers_dc[p.layer].z_min){ new_layer--; s = __fdividef(layers_dc[p.layer].z_min - p.z, p.dz); } //Check for upwards reflection/transmission & calculate new s
		if (p.z + s*p.dz>layers_dc[p.layer].z_max){ new_layer++; s = __fdividef(layers_dc[p.layer].z_max - p.z, p.dz); } //Check for downward reflection/transmission

		p.x += p.dx*s;
		p.y += p.dy*s;
		p.z += p.dz*s;

		if (p.z>layers_dc[p.layer].z_max)p.z = layers_dc[p.layer].z_max;//needed?
		if (p.z<layers_dc[p.layer].z_min)p.z = layers_dc[p.layer].z_min;//needed?

		if (new_layer != p.layer)
		{
			// set the remaining step length to 0
			s = 0.0f;

			if (Reflect(&p, new_layer, &x, &a) == 0u)//Check for reflection
			{ // Photon is transmitted
				if (new_layer == 0)
				{ //Diffuse reflectance
					index = __float2int_rz(acosf(-p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Rd_ra[index], p.weight);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
				if (new_layer > *n_layers_dc)
				{	//Transmitted
					index = __float2int_rz(acosf(p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Tt_ra[index], p.weight);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
			}
		}

		//w=0;

		if (s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered)
			w_temp = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			p.weight -= w_temp;


			//w = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			//p.weight -= w;

			if (ignoreAdetection == 0) // Evaluated at compiletime!
			{
				index = (min(__float2int_rz(__fdividef(p.z, det_dc[0].dz)), (int)det_dc[0].nz - 1)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1));
				if (index == index_old)
				{
					w += w_temp;
					//p.weight -= __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight)); 
				}
				else// if(w!=0)
				{
					AtomicAddULL(&DeviceMem.A_rz[index_old], w);
					index_old = index;
					w = w_temp;
				}

			}

			Spin(&p, &x, &a, layers_dc[p.layer].g);
		}




		if (!PhotonSurvive(&p, &x, &a)) // Check if photons survives or not
		{
			if (atomicAdd(DeviceMem.num_terminated_photons, 1u) < (*num_photons_dc))
			{	// Ok to launch another photon
				LaunchPhoton(&p);//Launch a new photon
			}
			else
			{	// No more photons should be launched. 
				DeviceMem.thread_active[DataPos] = 0u; // Set thread to inactive
				ii = NUMSTEPS_GPU;				// Exit main loop
			}

		}
	}//end main for loop!
	if (ignoreAdetection == 1 && w != 0)
		AtomicAddULL(&DeviceMem.A_rz[index_old], w);

	__syncthreads();//necessary?

	//save the state of the MC simulation in global memory before exiting
	DeviceMem.p[DataPos] = p;	//This one is incoherent!!!
	DeviceMem.x[DataPos] = x; //this one also seems to be coherent


}//end MCd
template <int ignoreAdetection> __global__ void CalcMCGPU(MemStruct DeviceMem)
{
	//Block index
	int bx = blockIdx.x;

	//Thread index
	int tx = threadIdx.x;


	//First element processed by the block
	int begin = blockDim.x*bx;
	if (DeviceMem.thread_active[begin + tx] == 65535){
		return;
	}
	if (DeviceMem.thread_active[begin + tx]){
		auto temp = atomicAdd(DeviceMem.num_terminated_photons, 1ul);
		if (temp > *num_photons_dc){
			DeviceMem.thread_active[begin + tx] = 65535;
			return;
		}
		DeviceMem.thread_active[begin + tx] = 0;
	}

	DeviceMem.thread_active[begin + tx] = 0;
	unsigned long long int x = DeviceMem.x[begin + tx];	//coherent
	unsigned int a = DeviceMem.a[begin + tx];			//coherent
	dsh_sPhoton[tx] = DeviceMem.p[begin + tx];
	float s; //step length
	

	unsigned int index, index_old;
	index_old = 0;
	unsigned long long w,w_temp;
	w = 0;

	int new_layer;

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;


	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
	{
		// Rand Make
		// �����ӂ�o�Ȃ����m�F
		if (layers_dc[dsh_sPhoton[tx].layer].mutr != FLT_MAX){
			// �����ɂ�鋗������
			s = -__logf(rand_MWC_oc(&x, &a))*layers_dc[dsh_sPhoton[tx].layer].mutr;	//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		}else{
			// �ꎞ�I��100 cm���
			s = 100.0f;															//temporary, say the step in glass is 100 cm.
		}
		// Hop_Drop() mcml_go
		//Check for layer transitions and in case, calculate s
		new_layer = dsh_sPhoton[tx].layer;
		// ���݂̃��C���[������Ɉړ����Ă邩���m�F
		if (dsh_sPhoton[tx].z + s*dsh_sPhoton[tx].dz<layers_dc[dsh_sPhoton[tx].layer].z_min){
			new_layer--; 
			s = __fdividef(layers_dc[dsh_sPhoton[tx].layer].z_min - dsh_sPhoton[tx].z, dsh_sPhoton[tx].dz); 
		} //Check for upwards reflection/transmission & calculate new s
		// ���݂̃��C���[�������Ɉړ����Ă邩���m�F
		if (dsh_sPhoton[tx].z + s*dsh_sPhoton[tx].dz>layers_dc[dsh_sPhoton[tx].layer].z_max){
			new_layer++;
			s = __fdividef(layers_dc[dsh_sPhoton[tx].layer].z_max - dsh_sPhoton[tx].z, dsh_sPhoton[tx].dz); 
		} //Check for downward reflection/transmission

		// �ʒu����
		dsh_sPhoton[tx].x += dsh_sPhoton[tx].dx*s;
		dsh_sPhoton[tx].y += dsh_sPhoton[tx].dy*s;
		dsh_sPhoton[tx].z += dsh_sPhoton[tx].dz*s;
//		Hop(&dsh_sPhoton[tx],s);
		if (dsh_sPhoton[tx].z > layers_dc[dsh_sPhoton[tx].layer].z_max){
			dsh_sPhoton[tx].z = layers_dc[dsh_sPhoton[tx].layer].z_max;//needed?
		}
		if (dsh_sPhoton[tx].z < layers_dc[dsh_sPhoton[tx].layer].z_min){
			dsh_sPhoton[tx].z = layers_dc[dsh_sPhoton[tx].layer].z_min;//needed?
		}
		//�@���C���[�ω����Ă����ꍇ

		if (new_layer != dsh_sPhoton[tx].layer)
		{
			// set the remaining step length to 0
			s = 0.0f;

			// ���˂��邩�m�F
			if (Reflect(&dsh_sPhoton[tx], new_layer, &x, &a) == 0u)//Check for reflection
			{ 
				// Photon is transmitted�@���q���`�B
				if (new_layer == 0)
				{	// Diffuse reflectance�@�g�U����
					// __float2int_rz �E�E�Efloat  => int�@�ւ̌^�ϊ�(�����_�؂�̂āH)
//					index = __float2int_rz(acosf(-dsh_sPhoton[tx].dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					index = __float2int_rz(__fdividef(acosf(-dsh_sPhoton[tx].dz) , (PI / det_dc[0].na)))*det_dc[0].nr + min(__float2int_rz(__fdividef(__fsqrt_rz(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Rd_ra[index], dsh_sPhoton[tx].weight);
					dsh_sPhoton[tx].weight = 0;
//					RecordR(dsh_sPhoton[tx]->rr, DeviceMem.In_Ptr, &dsh_sPhoton[tx], DeviceMem.Out_Ptr);//r���ǂ�����Ď����Ă���΂����̂�
					RemodelRecordR(DeviceMem, &dsh_sPhoton[tx]);//r���ǂ�����Ď����Ă���΂����̂�
				}
				if (new_layer > *n_layers_dc)
				{	//Transmitted�@����
					index = __float2int_rz(__fdividef(acosf(dsh_sPhoton[tx].dz), (PI / det_dc[0].na)))*det_dc[0].nr + min(__float2int_rz(__fdividef(__fsqrt_rz(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Tt_ra[index], dsh_sPhoton[tx].weight);
					dsh_sPhoton[tx].weight = 0;
				}
			}
		}
		//w=0;
		if (s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered) ���q�̎��ʌ���
			w_temp = __float2uint_rn(layers_dc[dsh_sPhoton[tx].layer].mua*layers_dc[dsh_sPhoton[tx].layer].mutr*__uint2float_rn(dsh_sPhoton[tx].weight));
			dsh_sPhoton[tx].weight -= w_temp;

			if (ignoreAdetection == 0) // Evaluated at compiletime!
			{
				index = (min(__float2int_rz(__fdividef(dsh_sPhoton[tx].z, det_dc[0].dz)), (int)det_dc[0].nz - 1)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1));
				if (index == index_old)
				{
					w += w_temp;
					//sharedp.weight -= __float2uint_rn(layers_dc[sharedp.layer].mua*layers_dc[sharedp.layer].mutr*__uint2float_rn(sharedp.weight)); 
				}
				else// if(w!=0)
				{
					AtomicAddULL(&DeviceMem.A_rz[index_old], w);
					index_old = index;
					w = w_temp;
				}

			}
			// �p�x�v�Z
			Spin(&dsh_sPhoton[tx], &x, &a, layers_dc[dsh_sPhoton[tx].layer].g);
		}



		if (!PhotonSurvive(&dsh_sPhoton[tx], &x, &a)) // Check if photons survives or not
		{
			DeviceMem.thread_active[begin + tx] = 1;
			LaunchPhoton(&dsh_sPhoton[tx]);
			break;
		}
	}//end main for loop!
	if (ignoreAdetection == 1 && w != 0)
		AtomicAddULL(&DeviceMem.A_rz[index_old], w);

	__syncthreads();//necessary?

	DeviceMem.x[begin + tx] = x; //this one also seems to be coherent
	DeviceMem.p[begin + tx] = dsh_sPhoton[tx]; //this one also seems to be coherent


}//end MCd

__device__  void LaunchPhoton(LayerStruct  * Layerspecs_Ptr,
	PhotonStruct* p,
	OutStruct    * Out_Ptr,
	InputStruct  * In_Ptr)
{
	// We are currently not using the RNG but might do later
	//float input_fibre_radius = 0.03;//[cm]
	//p->x=input_fibre_radius*sqrtf(rand_MWC_co(x,a));

	p->dead = 0;
	p->layer = 1;
	p->s = 0;
	p->sleft = 0;

	p->x = 0.0f;
	p->y = 0.0f;
	p->z = 0.0f;
	p->dx = 0.0f;
	p->dy = 0.0f;
	p->dz = 1.0f;

	p->layer = 1;
	p->weight = *start_weight_dc; //specular reflection!

}
__device__  void LaunchPhoton(PhotonStruct* p)
{

	p->dead = 0;
	p->layer = 1;
	p->s = 0;
	p->sleft = 0;

	p->x = 0.0f;
	p->y = 0.0f;
	p->z = 0.0f;
	p->dx = 0.0f;
	p->dy = 0.0f;
	p->dz = 1.0f;

	p->layer = 1;
	p->weight = *start_weight_dc; //specular reflection!

}

__global__ void LaunchPhoton_Global(PhotonStruct* pd)
{
	unsigned long long PosData = blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x;
	//First element processed by the block
	if (PosData < num_photons_dc[0]){
	
		//	 LaunchPhoton(&pd[PosData], d_x[PosData], d_a[PosData]);
		pd[PosData].dx = 0.0f;
		pd[PosData].dy = 0.0f;
		pd[PosData].dz = 1.0f;
		pd[PosData].x	= 0.0f;
		pd[PosData].y	= 0.0f;
		pd[PosData].z	= 0.0;
		pd[PosData].layer	= 1;
		pd[PosData].Index = PosData;
		pd[PosData].weight	= (unsigned int)*start_weight_dc;

		//DeviceMem->p[begin + tx] = p;//incoherent!?
		
	}
	return;
}
__global__ void SetRandpram(hiprandState* devState){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/* Each thread gets same seed, a different sequence number,
	no offset */
	hiprand_init(*dc_Seed, id, 0, &devState[id]);
}

__global__ void InitRng(MemStruct devMem,hiprandState* RndMakerglobal){
	hiprandState RndMakerLocal;
	RndMakerLocal = RndMakerglobal[threadIdx.x];
	unsigned long long* X = devMem.x;
	unsigned int* A = devMem.a;
	unsigned long long un64PosData = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int TmpRndH, TmpRndL;
	TmpRndH = hiprand(&RndMakerLocal);
	__syncthreads();
	TmpRndL = hiprand(&RndMakerLocal);
	unsigned long long TmpRAXH = ((unsigned long long)TmpRndH) << 32;
	X[un64PosData] = TmpRAXH | (unsigned long long)TmpRndL;
	__syncthreads();
	TmpRndL = hiprand(&RndMakerLocal);
	A[un64PosData] = TmpRndL;
	un64PosData += NUM_DIV_MAKE_RAND*NUM_THREADS_PER_BLOCK_MAKE_RAND;


		

	return;
}
__device__ double SpinTheta(unsigned long long int* x, unsigned int *a, double g){
	double cost;

	if (g == 0.0)
		cost = 2 * rand_MWC_co(x, a) - 1;
	else {
		double temp = (1 - g*g) / (1 - g + 2 * g*rand_MWC_co(x, a));
		cost = (1 + g*g - temp*temp) / (2 * g);
		if (cost < -1) cost = -1;
		else if (cost > 1) cost = 1;
	}
	return(cost);
}
__device__ void Hop(PhotonStruct* p,float s){
	p->x = s*p->dx;
	p->y = s*p->dy;
	p->z = s*p->dz;
}
__device__ void Spin(PhotonStruct* p, unsigned long long int* x, unsigned int *a, float g)
{
	float cost, sint;	// cosine and sine of the 
	// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
	// azimuthal angle psi. 
	float temp=2.1;

	float tempdir = p->dx;
	// Open CUDA Code
	//This is more efficient for g!=0 but of course less efficient for g==0
	//	temp = __fdividef((1.0f - (g)*(g)), (1.0f - (g)+2.0f*(g)*rand_MWC_co(x, a)));//Should be close close????!!!!!
	//	cost = __fdividef((1.0f + (g)*(g)-temp*temp), (2.0f*(g)));
	//	if (g == 0.0f)
	//		cost = 2.0f*rand_MWC_co(x, a) - 1.0f;//Should be close close??!!!!!

	// MIYAHIRA mcml SpinTheta()
	cost = SpinTheta(x,a,g);

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rand_MWC_co(x, a), &sinp, &cosp);// spin psi [0-2*PI)

	temp = sqrtf(1.0f - p->dz*p->dz);

	if (temp == 0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost, p->dz*cost);	// copysign(a,b)==  a*SIGN(b) 
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp), temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp), temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp = rsqrtf(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;

}// end Spin
__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, unsigned long long* x, unsigned int* a)
{
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)


	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);

	if (n1 == n2)//refraction index matching automatic transmission and no direction change
	{
		p->layer = new_layer;
		return 0u;
	}

	if (n1>n2 && n2*n2<n1*n1*(1 - cos_angle_i*cos_angle_i))//total internal reflection, no layer change but z-direction mirroring
	{
		p->dz *= -1.0f;
		return 1u;
	}

	if (cos_angle_i == 1.0f)//normal incident
	{
		r = __fdividef((n1 - n2), (n1 + n2));
		if (rand_MWC_co(x, a) <= r*r)
		{
			//reflection, no layer change but z-direction mirroring
			p->dz *= -1.0f;
			return 1u;
		}
		else
		{	//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
		}
	}
	else
	{
		//long and boring calculations of r
		float sinangle_i = sqrtf(1.0f - p->dz*p->dz);
		float sinangle_e = n1/n2*sinangle_i;
		float cosangle_e = sqrtf(1.0f - sinangle_e*sinangle_e);

		float cossumangle = (p->dz*cosangle_e) - sinangle_i*sinangle_e;
		float cosdiffangle = (p->dz*cosangle_e) + sinangle_i*sinangle_e;
		float sinsumangle = sinangle_i*cosangle_e + (p->z*sinangle_e);
		float sindiffangle = sinangle_i*cosangle_e - (p->z*sinangle_e);

		r = 0.5*sindiffangle*sindiffangle*__fdividef((cosdiffangle*cosdiffangle + cossumangle*cossumangle), (sinsumangle*sinsumangle*cosdiffangle*cosdiffangle));

	}
	//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
	// save a few multiplications, calculate cos_angle_i^2;
	//float e = __fdividef(n1*n1, n2*n2)*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
	//r = 2 * sqrtf((1.0f - cos_angle_i*cos_angle_i)*(1.0f - e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
	//e = e + (cos_angle_i*cos_angle_i)*(1.0f - 2.0f*e);//Update the value of e
	//r = e*__fdividef((1.0f - e - r), ((1.0f - e + r)*(e + r)));//Calculate r	

	if (rand_MWC_co(x, a) <= r)
	{
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		return 1u;
	}
	else
	{
		// Transmission, update layer and direction
		r = __fdividef(n1, n2);
		float e = r*r*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1 - e), p->dz);
		p->layer = new_layer;
		return 0u;
	}
	p->rr = r;
}
__device__ unsigned int PhotonSurvive(PhotonStruct* p, unsigned long long* x, unsigned int* a)
{	//Calculate wether the photon survives (returns 1) or dies (returns 0)

	if (p->weight>WEIGHTI) return 1u; // No roulette needed
	if (p->weight == 0u) return 0u;	// Photon has exited slab, i.e. kill the photon

	if (rand_MWC_co(x, a)<CHANCE)
	{
		p->weight = __float2uint_rn(__fdividef((float)p->weight, CHANCE));
		return 1u;
	}

	//else
	return 0u;
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned long long add)
{
	if (atomicAdd((unsigned long long*)address, add) + add<add)
		atomicAdd(((unsigned long long*)address) + 1, 1u);
}
__device__ float rand_MWC_co(unsigned long long* x, unsigned int* a)
{
	float temp = 0.0;
	//Generate a random number [0,1)
	*x = (*x & 0xffffffffull)*(*a) + (*x >> 32);
	temp = __fdividef(__uint2float_rz((unsigned int)(*x)), (float)0x100000000);// The typecast will truncate the x so that it is 0<=x<(2^32-1),__uint2float_rz ensures a round towards zero since 32-bit floating point cannot represent all integers that large. Dividing by 2^32 will hence yield [0,1)
	return temp;
}//end __device__ rand_MWC_co
__device__ float rand_MWC_oc(unsigned long long* x, unsigned int* a)
{
	//Generate a random number (0,1]
	return 1.0f - rand_MWC_co(x, a);
}//end __device__ rand_MWC_oc


cCUDAMCML::cCUDAMCML(){

}
cCUDAMCML::~cCUDAMCML(){
}
void cCUDAMCML::RunOldCarnel(){
	dim3 dimGrid(NUM_GRID_PER_BLOCK);
	dim3 dimBlock(NUM_THREADS_PER_BLOCK);
	unsigned int threads_active_total = 1;
	int i = 0;
	while (*m_sHostMem.num_terminated_photons < m_simulations->number_of_photons)
	{
		i++;
		//run the kernel
		if (m_simulations->ignoreAdetection == 1){
			MCd<1> << <dimGrid, dimBlock >> >(m_sDeviceMem);
		}
		else{
			MCd<0> << <dimGrid, dimBlock >> >(m_sDeviceMem);
		}
		hipDeviceSynchronize(); // Wait for all threads to finish
		hipError_t cudastat = hipGetLastError(); // Check if there was an error

		// Copy thread_active from device to host
		hipMemcpy(m_sHostMem.thread_active, m_sDeviceMem.thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyDeviceToHost);
		threads_active_total = 0;
		for (int ii = 0; ii < NUM_THREADS; ii++){
			threads_active_total += m_sHostMem.thread_active[ii];
		}

		hipMemcpy(m_sHostMem.num_terminated_photons, m_sDeviceMem.num_terminated_photons, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(m_sHostMem.p, m_sDeviceMem.p, sizeof(PhotonStruct)*m_simulations->number_of_photons, hipMemcpyDeviceToHost);
	}
	CopyDeviceToHostMem(&m_sHostMem, &m_sDeviceMem, m_simulations);
}
// �v�Z�̒���
int cCUDAMCML::MakeRandTableDev(){
	hiprandState *devStates;
	hipError_t  cudastat;
	dim3 dimNumBlockRand(NUM_GRID_PER_BLOCK);
	dim3 dimNumThreadRand(NUM_THREADS_PER_BLOCK);
	hipMalloc((void **)&devStates, NUM_THREADS * sizeof(hiprandState));
	// �V�[�h�C�����l�Ƃ��ėp���闐���z��̍쐬
	// MCML�̗��������ɗ��p�ł��Ȃ��@�ˁ@���C�u�����̓s����C�X���b�h������������邽��
	SetRandpram << < dimNumBlockRand, dimNumThreadRand >> > (devStates);
	cudastat = hipGetLastError();	// Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_RND_;
	}
	hipDeviceSynchronize();
	InitRng << < dimNumBlockRand, dimNumThreadRand >> > (m_sDeviceMem,devStates);
	// ���ؗp
	hipMemcpy(m_sHostMem.a, m_sDeviceMem.a, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(m_sHostMem.x, m_sDeviceMem.x, NUM_THREADS * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	cudastat = hipGetLastError();	// Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_RND_;
	}
	return 0;
}
int cCUDAMCML::InitPhoton(){


	dim3 dimNumBlock(19);
	dim3 dimNumThread(NUM_THREADS_PER_BLOCK);

	LaunchPhoton_Global << < dimNumBlock, dimNumThread >> > (m_sDeviceMem.p);
	hipError_t cudastat = hipGetLastError();	// Check if there was an error

	if (cudastat){
		return _ERR_GPU_SIM_LANCH_PHOTON_;
	}
	return 0;
}
int cCUDAMCML::DoOneSimulation(SimulationStruct* simulation)
{

	unsigned int threads_active_total = simulation->number_of_photons;
	unsigned int i;

	hipError_t cudastat;

	int STAT = 0;
	// Start the clock

	dim3 dimNumBlock(NUM_GRID_PER_BLOCK);
	dim3 dimNumThread(NUM_THREADS_PER_BLOCK);
	int TotalP = 0;
	while (TotalP<simulation->number_of_photons){
		//run the kernel
		if (simulation->ignoreAdetection == 1){
			CalcMCGPU<1> << < dimNumBlock, dimNumThread >> >(m_sDeviceMem);
	
		}
		else{
			CalcMCGPU<0> << < dimNumBlock, dimNumThread >> >(m_sDeviceMem);
	
		}
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat){
			return _ERR_GPU_SIM_MCML_;
		}
		// ���ؗp
		cudastat = hipMemcpy(m_sHostMem.p, m_sDeviceMem.p, NUM_THREADS * sizeof(PhotonStruct), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.thread_active, m_sDeviceMem.thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.num_terminated_photons, m_sDeviceMem.num_terminated_photons,sizeof(unsigned int), hipMemcpyDeviceToHost);
		//	hipDeviceSynchronize();		// Wait for all threads to finish
		// 
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat){
			return _ERR_GPU_SIM_MEMCPY_;
		}
		for (int i = 0; i < NUM_THREADS; i++){
			if (m_sHostMem.thread_active[i] != 65535){
				TotalP += m_sHostMem.thread_active[i];
			}
		}
	}

	cudastat = hipGetLastError(); // Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_ANOTHER_;
	}


	CopyDeviceToHostMem(&m_sHostMem, &m_sDeviceMem, simulation);
	return _SUCCESS_GPU_SIM_;

}
int cCUDAMCML::InitMallocMem(SimulationStruct* sim){
	unsigned int State = 0;
	hipError_t tmp;

	tmp = hipMalloc((void**)&m_sDeviceMem, sizeof(MemStruct));
	if (tmp != hipSuccess) {
		State |= 0x01;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.p, (NUM_THREADS)*sizeof(PhotonStruct));
	if (tmp != hipSuccess) {
		State |= 0x01;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.x, (NUM_THREADS)*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x02;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.a, (NUM_THREADS)*sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x04;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.In_Ptr, sizeof(InputStruct));
	if (tmp != hipSuccess) {
		State |= 0x08;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.Out_Ptr, sizeof(OutStruct));
	if (tmp != hipSuccess) {
		State |= 0x10;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.thread_active, (NUM_THREADS)*sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.num_terminated_photons, sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x40;
	}
	int rz_size = sim->det.nr*sim->det.nz;
	int ra_size = sim->det.nr*sim->det.na;
	tmp = hipMalloc((void**)&m_sDeviceMem.A_rz, rz_size *sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x80;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.Rd_ra, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x100;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.Tt_ra, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x200;
	}
	tmp = hipMalloc((void**)&m_Out.Rd_ra, ra_size*sizeof(double));
	if (tmp != hipSuccess) {
		State |= 0x400;
	}
	tmp = hipMalloc((void**)&m_Out.Rd_p, ra_size*sizeof(double));
	if (tmp != hipSuccess) {
		State |= 0x800;
	}

	// Allocate p on the device!!
	// Allocate A_rz on host and device
	m_sHostMem.p = new PhotonStruct			[NUM_THREADS];
	m_sHostMem.x = new unsigned long long	[NUM_THREADS];
	m_sHostMem.a = new unsigned int			[NUM_THREADS];
	if ((m_sHostMem.x != NULL) && (m_sHostMem.a != NULL)){
		State |= 0x00200000;
	}
	m_sHostMem.A_rz = new unsigned long long [rz_size];
	if (m_sHostMem.A_rz == NULL){
		State |= 0x00010000;
	}
	m_sHostMem.Rd_ra = new unsigned long long[ra_size];
	if (m_sHostMem.Rd_ra == NULL){
		State |= 0x00020000;
	}
	m_sHostMem.Tt_ra = new unsigned long long[ra_size];
	if (m_sHostMem.Tt_ra == NULL){
		State |= 0x00040000;
	}
	// Allocate thread_active on the device and host
	m_sHostMem.thread_active = new unsigned int[sim->number_of_photons];
	if (m_sHostMem.thread_active == NULL){
		State |= 0x00080000;
	}

	m_sHostMem.num_terminated_photons = new unsigned int[1];
	if (m_sHostMem.num_terminated_photons == NULL){
		State |= 0x00100000;
	}
	*m_sHostMem.num_terminated_photons = 0;

	m_sHostMem.Out_Ptr->Rd_ra = new double[ra_size];
	if (m_sHostMem.Out_Ptr->Rd_ra == NULL){
		State |= 0x00200000;
	}
	m_sHostMem.Out_Ptr->Rd_p = new double[rz_size];
	if (m_sHostMem.Out_Ptr->Rd_p == NULL){
		State |= 0x00400000;
	}
	m_Out.OPL = new double,sizeof(double);
	if (m_Out.OPL == NULL){
		State |= 0x00800000;
	}
	m_sHostMem.Out_Ptr->L = new double ,sizeof(double);
	if (m_Out.L == NULL){
		State |= 0x001000000;
	}
	m_sHostMem.Out_Ptr->opl = new double ,sizeof (double);
	if (m_sHostMem.Out_Ptr->Rd_p == NULL){
		State |= 0x02000000;
	}


	return State;
}
void cCUDAMCML::CopyDeviceToHostMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{ //Copy data from Device to Host memory

	int rz_size = sim->det.nr*sim->det.nz;
	int ra_size = sim->det.nr*sim->det.na;

	// Copy A_rz, Rd_ra and Tt_ra
	hipMemcpy(HostMem->A_rz, DeviceMem->A_rz, rz_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);
	hipMemcpy(HostMem->Rd_ra, DeviceMem->Rd_ra, ra_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);
	hipMemcpy(HostMem->Tt_ra, DeviceMem->Tt_ra, ra_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);

	//Also copy the state of the RNG's
	hipMemcpy(HostMem->p, DeviceMem->p, NUM_THREADS *sizeof(PhotonStruct), hipMemcpyDeviceToHost);

	return ;
}
int cCUDAMCML::CopyHostToDeviceMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim){
	// Allocate x and a on the device (For MWC RNG)
	hipError_t tmp;
	int State = 0;
	tmp = hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS*sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x40;
	}


	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x80;
	}
	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x200;
	}
	tmp = hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x800;
	}
	return State;
}
int cCUDAMCML::InitDCMem(SimulationStruct* sim)
{
	hipError_t tmp;
	int State = 0;
	// Copy det-data to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(det_dc), &(sim->det), sizeof(DetStruct));
	if (tmp != hipSuccess) {
		State |= 0x1;
	}

	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc), &(sim->n_layers), sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x2;
	}

	// Copy start_weight_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc), &(sim->start_weight), sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x4;
	}

	// Copy layer data to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(layers_dc), sim->layers, (sim->n_layers + 2)*sizeof(LayerStruct));
	if (tmp != hipSuccess) {
		State |= 0x8;
	}

	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc), &(sim->number_of_photons), sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x10;
	}
	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(dc_Seed), &(sim->Seed), sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}
	


	return State;

}
int cCUDAMCML::InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{
	int rz_size, ra_size;
	hipError_t tmp;
	rz_size = sim->det.nr*sim->det.nz;
	ra_size = sim->det.nr*sim->det.na;
	int Total = 0;

	// Allocate p on the device!!
	hipMalloc((void**)&DeviceMem->p, NUM_THREADS*sizeof(PhotonStruct));
	Total += NUM_THREADS*sizeof(PhotonStruct);
	// Allocate A_rz on host and device
	HostMem->A_rz = new(unsigned long long)(rz_size*sizeof(unsigned long long));
	if (HostMem->A_rz == NULL){ 
		printf("Error allocating HostMem->A_rz"); 
		exit(1); 
	}
	hipMalloc((void**)&DeviceMem->A_rz, rz_size*sizeof(unsigned long long));
	Total += rz_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->A_rz, 0, rz_size*sizeof(unsigned long long));

	// Allocate Rd_ra on host and device
	HostMem->Rd_ra = new(unsigned long long)(ra_size*sizeof(unsigned long long));
	if (HostMem->Rd_ra == NULL){ 
		printf("Error allocating HostMem->Rd_ra"); 
		exit(1);
	}
	hipMalloc((void**)&DeviceMem->Rd_ra, ra_size*sizeof(unsigned long long));
	Total += ra_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->Rd_ra, 0, ra_size*sizeof(unsigned long long));

	// Allocate Tt_ra on host and device
	HostMem->Tt_ra = new(unsigned long long)(ra_size*sizeof(unsigned long long));
	if (HostMem->Tt_ra == NULL){ printf("Error allocating HostMem->Tt_ra"); exit(1); }
	hipMalloc((void**)&DeviceMem->Tt_ra, ra_size*sizeof(unsigned long long));
	Total += ra_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->Tt_ra, 0, ra_size*sizeof(unsigned long long));


	// Allocate x and a on the device (For MWC RNG)
	hipMalloc((void**)&DeviceMem->x, NUM_THREADS*sizeof(unsigned long long));
	Total += NUM_THREADS*sizeof(unsigned long long);

	hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS*sizeof(unsigned long long), hipMemcpyHostToDevice);
	tmp = hipMalloc((void**)&DeviceMem->a, NUM_THREADS*sizeof(unsigned int));
	Total += NUM_THREADS*sizeof(unsigned long long);

	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);


	// Allocate thread_active on the device and host
	HostMem->thread_active = new unsigned int (NUM_THREADS*sizeof(unsigned int));
	if (HostMem->thread_active == NULL){ printf("Error allocating HostMem->thread_active"); exit(1); }
	for (int i = 0; i < NUM_THREADS; i++){
		HostMem->thread_active[i] = 1u;
	}

	Total += NUM_THREADS*sizeof(unsigned int);
	tmp = hipMalloc((void**)&(DeviceMem->thread_active), NUM_THREADS*sizeof(unsigned int));

	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);


	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = new(unsigned int)(sizeof(unsigned int));
	if (HostMem->num_terminated_photons == NULL){ printf("Error allocating HostMem->num_terminated_photons"); exit(1); }
	*HostMem->num_terminated_photons = 0;

	hipMalloc((void**)&DeviceMem->num_terminated_photons, sizeof(unsigned int));
	hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);

	return 1;
}
int cCUDAMCML::InitContentsMem(SimulationStruct* sim)
{
	MemStruct* DeviceMem = &m_sDeviceMem;
	MemStruct* HostMem = &m_sHostMem;

	int State=0;
	int rz_size, ra_size;
	hipError_t tmp;
	rz_size = sim->det.nr*sim->det.nz;
	ra_size = sim->det.nr*sim->det.na;

	tmp = hipMemset(DeviceMem->A_rz, 0, rz_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x02;
	}

	tmp = hipMemset(DeviceMem->Rd_ra, 0, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x08;
	}

	tmp = hipMemset(DeviceMem->Tt_ra, 0, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}

	PhotonStruct TmpPS;

	HostMem->p->x = 0;
	HostMem->p->y = 0;
	HostMem->p->z = 0;
	HostMem->p->dx = 0;
	HostMem->p->dy = 0;
	HostMem->p->dz = 0;
	HostMem->p->weight = 0;
	HostMem->p->layer = 0;

	tmp = hipMemset(DeviceMem->p, 0, NUM_THREADS *sizeof(PhotonStruct));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}

	tmp = hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS *sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x40;
	}


	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x80;
	}

	for (int i = 0; i < sim->number_of_photons; i++){
		HostMem->thread_active[i] = 1u;
	}



	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x200;
	}



	tmp = hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x800;
	}

	return State;
}

void cCUDAMCML::FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{

	hipFree(DeviceMem->p);
	hipFree(DeviceMem->x);
	hipFree(DeviceMem->a);
	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);
	hipFree(DeviceMem->A_rz);
	hipFree(DeviceMem->Rd_ra);
	hipFree(DeviceMem->Tt_ra);
	hipFree(DeviceMem);
	delete[] HostMem->p;
	delete[] HostMem->x;
	delete[] HostMem->a;
	delete[] HostMem->thread_active;
	delete[] HostMem->num_terminated_photons;
	delete[] HostMem->Reserve;
	delete[] HostMem->A_rz;
	delete[] HostMem->Rd_ra;
	delete[] HostMem->Tt_ra;


}

void cCUDAMCML::FreeSimulationStruct(SimulationStruct* sim, int nRun)
{
	FreeMemStructs(&m_sHostMem, &m_sDeviceMem);
	for (int i = 0; i < nRun; i++){
		delete sim[i].layers;
	}
	delete[] sim;
	//hipDeviceReset();
}

void cCUDAMCML::FreeFailedSimStrct(SimulationStruct* Sim, int nRun)
{
	for (int i = 0; i < nRun; i++){
		delete Sim[i].layers;
	}
	delete[] Sim;

}


bool cCUDAMCML::CheckGPU(){
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0){
		return 0;
	}
	hipSetDevice(0);
	hipGetDeviceProperties(&m_sDevProp, 0);
	return 1;
}
void cCUDAMCML::InitGPUStat(){
	m_ProcessTime = 0;
	m_un64Membyte = 0;
	m_un64NumPhoton = 0;
	m_un64PrcsDataNum = 0;
	hipDeviceReset();
}
/***********************************************************
*	Record the photon weight exiting the first layer(uz<0),
*	no matter whether the layer is glass or not, to the
*	reflection array.
*
*	Update the photon weight as well.
****/
extern "C"{
	__host__ __device__ void RemodelRecordR(MemStruct  DeviceMem, PhotonStruct *p)
	{
		InputStruct *In_Ptr = DeviceMem.In_Ptr;
		OutStruct	*Out_Ptr = DeviceMem.Out_Ptr;
		double	Refl = p->rr;
		double x = p->x;
		double y = p->y;
		double dx = p->dx;
		double dy = p->dy;
		double dz = p->dz;
		double t;
		double r;
		double r1 = In_Ptr->r;
		double t1, t2, t3;
		short  it, ia;	/* index to r & angle. */
		double itd, iad;	/* LW 5/20/98. To avoid out of short range.*/
		short  nl = In_Ptr->num_layers;
		short	 l;
		int	 n = Out_Ptr->p1;
		int id;

		r = sqrt(x*x + y*y);

		if (r >= r1 && r <= (r1 + 0.1*r1))
		{
			if (y >= 0)
				t1 = atan2(y, x) * 180 / PI;
			else
				t1 = 360 + atan2(y, x) * 180 / PI;

			if (dy <= 0)
				t2 = atan2(-dy, -dx) * 180 / PI;
			else
				t2 = 360 + atan2(-dy, -dx) * 180 / PI;

			t3 = t2 - t1;
			if (t3 < 0)
				t = 360 + t3;
			else
				t = t3;

			itd = (short)(t / In_Ptr->dt);
			if (itd > In_Ptr->nr - 1) it = In_Ptr->nr - 1;
			else it = itd;

			iad = (short)(acos(-dz) * 180 / PI / In_Ptr->da);
			if (iad > In_Ptr->na - 1) ia = In_Ptr->na - 1;
			else ia = iad;

			Out_Ptr->Rd_ra[In_Ptr->nr*ia + it] += p->weight*(1.0 - Refl);		/* �e�V���p�E�e���ʊp�̌��q�E�F�C�g�̋L�^ */
			Out_Ptr->Rd_p[In_Ptr->nr*ia + it] += 1;							/* �e�V���p�E�e���ʊp�̌��q���̋L�^ */
			Out_Ptr->P += p->weight*(1.0 - Refl);

			for (l = 1; l <= nl; l++)
				Out_Ptr->L[l] += Out_Ptr->OPL[l] * p->weight*(1.0 - Refl);		/* ����G���A�ɓ��������q�̌��H���̋L�^ */

			Out_Ptr->p1 += 1;
		}

		p->weight *= Refl;
	}
	//__host__ __device__ void RecordR(double			Refl,	/* reflectance. */
	//	InputStruct  *	In_Ptr,
	//	PhotonStruct *	p,
	//	OutStruct *	Out_Ptr)
	//{
	//	double x = p->x;
	//	double y = p->y;
	//	double dx = p->dx;
	//	double dy = p->dy;
	//	double dz = p->dz;
	//	double t;
	//	double r;
	//	double r1 = In_Ptr->r;
	//	double t1, t2, t3;
	//	short  it, ia;	/* index to r & angle. */
	//	double itd, iad;	/* LW 5/20/98. To avoid out of short range.*/
	//	short  nl = In_Ptr->num_layers;
	//	short	 l;
	//	int	 n = Out_Ptr->p1;
	//
	//	r = sqrt(x*x + y*y);
	//
	//	if (r >= r1 && r <= (r1 + 0.1*r1))
	//	{
	//		if (y >= 0)
	//			t1 = atan2(y, x) * 180 / PI;
	//		else
	//			t1 = 360 + atan2(y, x) * 180 / PI;
	//
	//		if (dy <= 0)
	//			t2 = atan2(-dy, -dx) * 180 / PI;
	//		else
	//			t2 = 360 + atan2(-dy, -dx) * 180 / PI;
	//
	//		t3 = t2 - t1;
	//		if (t3<0)
	//			t = 360 + t3;
	//		else
	//			t = t3;
	//
	//		itd = (short)(t / In_Ptr->dt);
	//		if (itd>In_Ptr->nr - 1) it = In_Ptr->nr - 1;
	//		else it = itd;
	//
	//		iad = (short)(acos(-dz) * 180 / PI / In_Ptr->da);
	//		if (iad>In_Ptr->na - 1) ia = In_Ptr->na - 1;
	//		else ia = iad;
	//
	//		Out_Ptr->Rd_ra[it][ia] += p->weight*(1.0 - Refl);		/* �e�V���p�E�e���ʊp�̌��q�E�F�C�g�̋L�^ */
	//		Out_Ptr->Rd_p[it][ia] += 1;							/* �e�V���p�E�e���ʊp�̌��q���̋L�^ */
	//		Out_Ptr->P += p->weight*(1.0 - Refl);
	//
	//		for (l = 1; l <= nl; l++)
	//			Out_Ptr->L[l] += Out_Ptr->OPL[l] * p->weight*(1.0 - Refl);		/* ����G���A�ɓ��������q�̌��H���̋L�^ */
	//
	//		Out_Ptr->p1 += 1;
	//	}
	//
	//	p->weight *= Refl;
	//}
	__host__ __device__ void InitOutputData(MemStruct deviceMem, InputStruct In_Parm,
		OutStruct * Out_Ptr)
	{
		short nr = In_Parm.nr;
		short na = In_Parm.na;
		short nl = In_Parm.num_layers;
		/* remember to use nl+2 because of 2 for ambient. */

		if (nr <= 0 || na <= 0 || nl <= 0)
			//nrerror("Wrong grid parameters.\n");

			/* Init pure numbers. */
			Out_Ptr->Rsp = 0.0;

		/* Allocate the arrays and the matrices. */
		Out_Ptr->Rd_ra = AllocMatrix(0, nr - 1, 0, na - 1);
		Out_Ptr->Rd_p = AllocMatrix(0, nr - 1, 0, na - 1);

		Out_Ptr->OPL = AllocVector(0, nl + 1);
		Out_Ptr->L = AllocVector(0, nl + 1);
		Out_Ptr->opl = AllocVector(0, nl + 1);
	}
	void ReportResult(InputStruct In_Parm, OutStruct Out_Parm)
	{
		char time_report[STR_LEN];

		strcpy(time_report, " Simulation time of this run.");
		PunchTime(1, time_report);

		SumScaleResult(In_Parm, &Out_Parm);
		WriteResult(In_Parm, Out_Parm, time_report);
	}
	__host__ __device__ time_t PunchTime(char F, char *Msg)
	{
#if GNUCC
		return(0);
#else
		static clock_t ut0;	/* user time reference. */
		static time_t  rt0;	/* real time reference. */
		double secs;
		char s[STR_LEN];

		if (F == 0) {
			ut0 = clock();
			rt0 = time(NULL);
			return(0);
		}
		else if (F == 1)  {
			secs = (clock() - ut0) / (double)CLOCKS_PER_SEC;
			if (secs < 0) secs = 0;	/* clock() can overflow. */
			sprintf(s, "User time: %8.0lf sec = %8.2lf hr.  %s\n",
				secs, secs / 3600.0, Msg);
			puts(s);
			strcpy(Msg, s);
			return(difftime(time(NULL), rt0));
		}
		else if (F == 2) return(difftime(time(NULL), rt0));
		else return(0);
#endif
	}
	__host__ __device__ void SumScaleResult(InputStruct In_Parm, OutStruct * Out_Ptr)
	{
		CalOPL_SD(In_Parm, Out_Ptr);
	}
	__host__ void WriteResult(InputStruct In_Parm,
		OutStruct Out_Parm,
		char * TimeReport)
	{
		FILE *file;

		file = fopen(In_Parm.out_fname, "w");
		//	if (file == NULL) nrerror("Cannot open file to write.\n");

		if (toupper(In_Parm.out_fformat) == 'A')
			WriteVersion(file, "A1");
		else
			WriteVersion(file, "B1");

		fprintf(file, "# %s", TimeReport);
		fprintf(file, "\n");

		WriteInParm(file, In_Parm);
		/* reflectance, absorption, transmittance. */

		/* 1D arrays. */

		/* 2D arrays. */
		WriteRd_ra(file, In_Parm.nr, In_Parm.na, Out_Parm);
		WriteRd_p(file, In_Parm.nr, In_Parm.na, Out_Parm);
		WriteOPL(file, In_Parm.num_layers, Out_Parm);

		fclose(file);
	}
	__device__ __host__ void CalOPL_SD(InputStruct In_Parm, OutStruct * Out_Ptr)
	{
		short l;
		short	nl = In_Parm.num_layers;

		for (l = 1; l <= nl; l++)
			Out_Ptr->opl[l] = Out_Ptr->L[l] / Out_Ptr->P;		/* ��n�w�ɓ��������q�̌��H���̕��� */
	}

	__device__ __host__ double *AllocVector(short nl, short nh)
	{
		double *v;
		short i;

		v = (double *)malloc((unsigned)(nh - nl + 1)*sizeof(double));
		//	if (!v) nrerror("allocation failure in vector()");

		v -= nl;
		for (i = nl; i <= nh; i++) v[i] = 0.0;	/* init. */
		return v;
	}
	__host__ void WriteVersion(FILE *file, char *Version)
	{
		fprintf(file,
			"%s \t# Version number of the file format.\n\n",
			Version);
		fprintf(file, "####\n# Data categories include: \n");
		fprintf(file, "# InParm, RAT, \n");
		fprintf(file, "# Rd_ta, \n####\n\n");
	}
	/***********************************************************
	*	Write the input parameters to the file.
	****/
	void WriteInParm(FILE *file, InputStruct In_Parm)
	{
		short i;

		fprintf(file,
			"InParm \t\t\t# Input parameters. cm is used.\n");

		fprintf(file,
			"%s \tA\t\t# output file name, ASCII.\n",
			In_Parm.out_fname);
		fprintf(file,
			"%ld \t\t\t# No. of photons\n", In_Parm.num_photons);
		fprintf(file,
			"%.2lf \t\t\t# No. of SD distance\n", In_Parm.r);

		fprintf(file,
			"%G\t\t\t\t# dt [cm]\n", In_Parm.dt);
		fprintf(file, "%hd\t%hd\t\t# No. of dt, da.\n\n",
			In_Parm.nr, In_Parm.na);

		fprintf(file,
			"%hd\t\t\t\t\t# Number of layers\n",
			In_Parm.num_layers);
		fprintf(file,
			"#n\tmua\tmus\tg\td\t# One line for each layer\n");
		fprintf(file,
			"%G\t\t\t\t\t# n for medium above\n",
			In_Parm.layerspecs[0].n);
		for (i = 1; i <= In_Parm.num_layers; i++)  {
			LayerStruct s;
			s = In_Parm.layerspecs[i];
			fprintf(file, "%G\t%G\t%G\t%G\t%G\t# layer %hd\n",
				s.n, s.mua, s.mutr, s.g, s.z_max - s.z_min, i);
		}
		fprintf(file, "%G\t\t\t\t\t# n for medium below\n\n",
			In_Parm.layerspecs[i].n);
	}
	__host__ void WriteRd_ra(FILE * file,
		short Nr,
		short Na,
		OutStruct Out_Parm)
	{
		short it, ia;

		fprintf(file,
			"%s\n%s\n%s\n%s\n%s\n%s\n",	/* flag. */
			"# Rd[theta][angle]. [1/(cm2sr)].",
			"# Rd[0][0], [0][1],..[0][na-1]",
			"# Rd[1][0], [1][1],..[1][na-1]",
			"# ...",
			"# Rd[nt-1][0], [nt-1][1],..[nt-1][na-1]",
			"Rd_ta");

		for (it = 0; it < Nr; it++)
		{
			for (ia = 0; ia < Na; ia++)
			{
				fprintf(file, "%12.4E,", Out_Parm.Rd_ra[it + Nr*ia]);
				if ((it*Na + ia + 1) % 9 == 0) fprintf(file, "\n");
			}
		}
		fprintf(file, "\n");
	}

	/***********************************************************
	*	1 number each line.
	****/

	__host__ void WriteRd_p(FILE * file,
		short Nr,
		short Na,
		OutStruct Out_Parm)
	{
		short it, ia;

		fprintf(file,
			"%s\n%s\n%s\n%s\n%s\n%s\n",	/* flag. */
			"# Rd[theta][angle]. [1/(cm2sr)].",
			"# Rd[0][0], [0][1],..[0][na-1]",
			"# Rd[1][0], [1][1],..[1][na-1]",
			"# ...",
			"# Rd[nt-1][0], [nt-1][1],..[nt-1][na-1]",
			"Rd_p");

		for (it = 0; it < Nr; it++)
		{
			for (ia = 0; ia < Na; ia++)
			{
				fprintf(file, "%12.4E,", Out_Parm.Rd_p[it + ia*Nr]);
				if ((it*Na + ia + 1) % 9 == 0) fprintf(file, "\n");
			}
		}
		fprintf(file, "photon number");
		fprintf(file, "%ld\n", Out_Parm.p1);
		fprintf(file, "\n");
	}

	/***********************************************************
	*	1 number each line.
	****/
	__host__ void WriteOPL(FILE * file,
		short nl,
		OutStruct Out_Parm)
	{
		short l;

		for (l = 1; l <= nl; l++)
		{
			fprintf(file, "The %d layer\n", l);
			fprintf(file, "%12.4E\n", Out_Parm.opl[l]);	/* ���ό��H���̏������� */
			fprintf(file, "\n");
		}
	}
	__host__ __device__ double *AllocMatrix(short nrl, short nrh, short ncl, int nch)
	{
		long i, j;
		double *m;

		m = (double *)malloc((unsigned)(nrh - nrl + 1)*(nch - ncl + 1) *sizeof(double*));
		//if (!m) nrerror("allocation failure 1 in matrix()");
		//m -= nrl;

		//for (i = nrl; i <= nrh; i++) {
		//m[i] = (double *)malloc((unsigned)(nch - ncl + 1) *sizeof(double));
		//if (!m[i]) nrerror("allocation failure 2 in matrix()");
		//m[i] -= ncl;
		//}

		for (i = nrl; i <= nrh; i++)
			for (j = ncl; j <= nch; j++) m[i*(nrh - nrl + 1) + j] = 0.0;
		return m;
	}
}
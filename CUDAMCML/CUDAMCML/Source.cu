#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define _NVCC_
#include "CUDAMCML_GPGPU.h"
#include <hiprand/hiprand_kernel.h>
#include <fstream>

#define _ERR_GPU_SIM_RND_ 1
#define _ERR_GPU_SIM_MEMCPY_ 2
#define _ERR_GPU_SIM_LOOP_ 3
#define _ERR_GPU_SIM_LANCH_PHOTON_ 4
#define _ERR_GPU_SIM_MCML_ 5
#define _ERR_GPU_SIM_ANOTHER_ 0xFF
#define _SUCCESS_GPU_SIM_ 0
#define PARTIALREFLECTION 1     
#define GNUCC 0
/* 1=split photon, 0=statistical reflection. */

// MemStruct m_sDeviceMem;

__device__ __constant__ unsigned int num_photons_dc[1];
__device__ __constant__ unsigned int n_layers_dc[1];
__device__ __constant__ unsigned long long start_weight_dc[1];
__device__ __constant__ LayerStruct layers_dc[MAX_LAYERS];
__device__ __constant__ DetStruct det_dc[1];
__device__ __constant__ unsigned int dc_Seed[1];
__device__ unsigned int nInitRngLoop = 0;
__device__ unsigned int global[NUM_THREADS];

__shared__ PhotonStruct dsh_sPhoton[NUM_THREADS_PER_BLOCK];

unsigned int SimulationStruct::GetRaSize(){
	return det.nr*det.na;
}
unsigned int SimulationStruct::GetRzSize(){
	return det.nr*det.nz;
}

//__global__ void ReflectTest(MemStruct DeviceMem){
//	//Block index
//	int bx = blockIdx.x;
//
//	//Thread index
//	int tx = threadIdx.x;
//	int new_layer = 0;
//
//
//
//	//First element processed by the block
//	int begin = blockDim.x*bx;
//
//	PhotonStruct* p = &DeviceMem.p[begin + tx];
//	p->dead = 0;
//	p->s = 0;
//	p->sleft = 0;
//	
//	p->x = 0.0f;
//	p->y = 0.0f;
//	p->z = 0.001f;
//	p->dx = 0.2f;
//	p->dy = 0.3f;
//	p->dz = -0.9f+tx*0.001;
//	
//	p->dx = p->dx / sqrt(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
//	p->dy = p->dy / sqrt(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
//	p->dz = p->dz / sqrt(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
//	
//	p->layer = 1;
//	p->weight = *start_weight_dc;
//	//p->dead=Reflecta(p, 0, DeviceMem.x, DeviceMem.a);
//
//}

//
// MCML�v�Z�̖{��
//// 
//template <int ignoreAdetection> __global__ void MCd(MemStruct DeviceMem)
//{
//	//Block index
//	int bx = blockIdx.x;
//
//	//Thread index
//	int tx = threadIdx.x;
//
//
//	//First element processed by the block
//	int begin = blockDim.x*bx;
//
//
//
//	unsigned long long int x = DeviceMem.x[begin + tx];//coherent
//	unsigned int a = DeviceMem.a[begin + tx];//coherent
//
//	float s;	//step length
//	//p->s = s;   //PhotonStruct��s�Ƃ̓������K�v�H
//
//	unsigned long long index, w, index_old,DataPos;
//	index_old = 0;
//	w = 0;
//	unsigned int w_temp;
//	DataPos = *DeviceMem.num_terminated_photons;
//
//	PhotonStruct p = DeviceMem.p[begin + tx];
//	
//
//	int new_layer;
//
//	//First, make sure the thread (photon) is active
//	unsigned int ii = 0;
//	if (!DeviceMem.thread_active[begin + tx]){
//		ii = NUMSTEPS_GPU;
//	}
//
//	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
//	{
//		if (layers_dc[p.layer].mutr != FLT_MAX)
//			p.s = -__logf(rand_MWC_oc(&x, &a))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
//		else
//			p.s = 100.0f;//temporary, say the step in glass is 100 cm.
//
//		//Check for layer transitions and in case, calculate s
//		new_layer = p.layer;
//		if (p.z + s*p.dz<layers_dc[p.layer].z_min){ new_layer--; s = __fdividef(layers_dc[p.layer].z_min - p.z, p.dz); } //Check for upwards reflection/transmission & calculate new s
//		if (p.z + s*p.dz>layers_dc[p.layer].z_max){ new_layer++; s = __fdividef(layers_dc[p.layer].z_max - p.z, p.dz); } //Check for downward reflection/transmission
//
//		p.x += p.dx*s;
//		p.y += p.dy*s;
//		p.z += p.dz*s;
//
//		if (p.z>layers_dc[p.layer].z_max)p.z = layers_dc[p.layer].z_max;//needed?
//		if (p.z<layers_dc[p.layer].z_min)p.z = layers_dc[p.layer].z_min;//needed?
//
//		if (new_layer != p.layer)
//		{
//			// set the remaining step length to 0
//			s = 0.0f;
//
//			if (Reflect(&p, new_layer, &x, &a) == 0u)//Check for reflection
//			{ // Photon is transmitted
//				if (new_layer == 0)
//				{ //Diffuse reflectance
//					index = __float2int_rz(acosf(-p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
//					AtomicAddULL(&DeviceMem.Rd_ra[index], p.weight);
//					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
//				}
//				if (new_layer > *n_layers_dc)
//				{	//Transmitted
//					index = __float2int_rz(acosf(p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
//					AtomicAddULL(&DeviceMem.Tt_ra[index], p.weight);
//					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
//				}
//			}
//		}
//
//		//w=0;
//
//		if (s > 0.0f)
//		{
//			// Drop weight (apparently only when the photon is scattered)
//			w_temp = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
//			p.weight -= w_temp;
//
//
//			//w = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
//			//p.weight -= w;
//
//			if (ignoreAdetection == 0) // Evaluated at compiletime!
//			{
//				index = (min(__float2int_rz(__fdividef(p.z, det_dc[0].dz)), (int)det_dc[0].nz - 1)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), det_dc[0].dr)), (int)det_dc[0].nr - 1));
//				if (index == index_old)
//				{
//					w += w_temp;
//					//p.weight -= __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight)); 
//				}
//				else// if(w!=0)
//				{
//					AtomicAddULL(&DeviceMem.A_rz[index_old], w);
//					index_old = index;
//					w = w_temp;
//				}
//
//			}
//
//			Spin(&p, &x, &a, layers_dc[p.layer].g);
//		}
//
//
//
//
//		if (!PhotonSurvive(&p, &x, &a)) // Check if photons survives or not
//		{
//			if (atomicAdd(DeviceMem.num_terminated_photons, 1u) < (*num_photons_dc))
//			{	// Ok to launch another photon
//				LaunchPhoton(&p);//Launch a new photon
//			}
//			else
//			{	// No more photons should be launched. 
//				DeviceMem.thread_active[DataPos] = 0u; // Set thread to inactive
//				ii = NUMSTEPS_GPU;				// Exit main loop
//			}
//
//		}
//	}//end main for loop!
//	if (ignoreAdetection == 1 && w != 0)
//		AtomicAddULL(&DeviceMem.A_rz[index_old], w);
//
//	__syncthreads();//necessary?
//
//	//save the state of the MC simulation in global memory before exiting
//	DeviceMem.p[DataPos] = p;	//This one is incoherent!!!
//	DeviceMem.x[DataPos] = x; //this one also seems to be coherent
//
//
//}//end MCd
template <int ignoreAdetection> __global__ void CalcMCGPU(MemStruct DeviceMem)
{
	
	//Block index
	int bx = blockIdx.x;

	//Thread index
	int tx = threadIdx.x;


	//First element processed by the block
	int begin = blockDim.x*bx;
	if (DeviceMem.thread_active[begin + tx] == 65535){
		return;
	}
	if (DeviceMem.thread_active[begin + tx]){
		auto temp = atomicAdd(DeviceMem.num_terminated_photons, 1ul);
		if (temp > *num_photons_dc){
			DeviceMem.thread_active[begin + tx] = 65535;
			return;
		}
		DeviceMem.thread_active[begin + tx] = 0;
	}

	DeviceMem.thread_active[begin + tx] = 0;
	unsigned long long int x = DeviceMem.x[begin + tx];	//coherent
	unsigned int a = DeviceMem.a[begin + tx];			//coherent
	dsh_sPhoton[tx] = DeviceMem.p[begin + tx];
	
	

	unsigned int index, index_old;
	index_old = 0;
	unsigned long long w,w_temp;
	w = 0;

	int new_layer;

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;

	DeviceMem.check[begin + tx].c = 10.0;
	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
	{
		DeviceMem.check[begin + tx].c = 0.0;
		// Rand Make
		// �����ӂ�o�Ȃ����m�F
		if (layers_dc[dsh_sPhoton[tx].layer].mutr != FLT_MAX){
			// �����ɂ�鋗������
			dsh_sPhoton[tx].s = -__logf(rand_MWC_oc(&x, &a))*layers_dc[dsh_sPhoton[tx].layer].mutr;	//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		}else{
			// �ꎞ�I��100 cm���
			dsh_sPhoton[tx].s = 100.0f;															//temporary, say the step in glass is 100 cm.
		}
		DeviceMem.check[begin + tx].cc = dsh_sPhoton[tx].s;
		// Hop_Drop() mcml_go
		//Check for layer transitions and in case, calculate s
		new_layer = dsh_sPhoton[tx].layer;
		// ���݂̃��C���[������Ɉړ����Ă邩���m�F
		if (dsh_sPhoton[tx].z + dsh_sPhoton[tx].s*dsh_sPhoton[tx].dz<layers_dc[dsh_sPhoton[tx].layer].z_min){
			new_layer--; 
			dsh_sPhoton[tx].s = __fdividef(layers_dc[dsh_sPhoton[tx].layer].z_min - dsh_sPhoton[tx].z, dsh_sPhoton[tx].dz);
		} //Check for upwards reflection/transmission & calculate new s
		// ���݂̃��C���[�������Ɉړ����Ă邩���m�F
		if (dsh_sPhoton[tx].z + dsh_sPhoton[tx].s*dsh_sPhoton[tx].dz>layers_dc[dsh_sPhoton[tx].layer].z_max){
			new_layer++;
			dsh_sPhoton[tx].s = __fdividef(layers_dc[dsh_sPhoton[tx].layer].z_max - dsh_sPhoton[tx].z, dsh_sPhoton[tx].dz);
		} //Check for downward reflection/transmission

		// �ʒu����
		dsh_sPhoton[tx].x += dsh_sPhoton[tx].dx*dsh_sPhoton[tx].s;
		dsh_sPhoton[tx].y += dsh_sPhoton[tx].dy*dsh_sPhoton[tx].s;
		dsh_sPhoton[tx].z += dsh_sPhoton[tx].dz*dsh_sPhoton[tx].s;
//		Hop(&dsh_sPhoton[tx],s);
		if (dsh_sPhoton[tx].z > layers_dc[dsh_sPhoton[tx].layer].z_max){
			dsh_sPhoton[tx].z = layers_dc[dsh_sPhoton[tx].layer].z_max;//needed?
		}
		if (dsh_sPhoton[tx].z < layers_dc[dsh_sPhoton[tx].layer].z_min){
			dsh_sPhoton[tx].z = layers_dc[dsh_sPhoton[tx].layer].z_min;//needed?
		}
		
		if (new_layer != dsh_sPhoton[tx].layer)
		{
			// set the remaining step length to 0
			dsh_sPhoton[tx].s = 0.0f;
			
			DeviceMem.check[begin + tx].c = 1.0;
			
			// ���˂��邩�m�F
			DeviceMem.check[begin + tx].dz = dsh_sPhoton[tx].dz;
			DeviceMem.check[begin + tx].r = 0;
			unsigned int  returnvalue = 0;
			DeviceMem.check[begin + tx].r = Reflect(&dsh_sPhoton[tx], new_layer, &x, &a, &returnvalue);
			//Check for reflection.0�͓��߁C1�͔���
			//DeviceMem.check[begin + tx].r = global[begin + tx];
			if (DeviceMem.check[begin + tx].r == 0u)
			{ 
				
				DeviceMem.check[begin + tx].c = 2.0;
				// Photon is transmitted�@���q���`�B
				if (new_layer == 0)
				{	// Diffuse reflectance�@�g�U����
					// __float2int_rz �E�E�Efloat  => int�@�ւ̌^�ϊ�(�����_�؂�̂āH)
//					index = __float2int_rz(acosf(-dsh_sPhoton[tx].dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					index = __float2int_rz(__fdividef(acosf(-dsh_sPhoton[tx].dz) , (PI / det_dc[0].na)))*det_dc[0].nr + min(__float2int_rz(__fdividef(__fsqrt_rz(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Rd_ra[index], dsh_sPhoton[tx].weight);
					
//					RecordR(dsh_sPhoton[tx]->rr, DeviceMem.In_Ptr, &dsh_sPhoton[tx], DeviceMem.Out_Ptr);//r���ǂ�����Ď����Ă���΂����̂�
				
					DeviceMem.check[begin + tx].c = 3.0;
					RemodelRecordR(DeviceMem, &dsh_sPhoton[tx]);
					DeviceMem.check[begin + tx].w = dsh_sPhoton[tx].weight;
					
					dsh_sPhoton[tx].weight = 0;
				}
				if (new_layer > *n_layers_dc)
				{	//Transmitted�@����
					index = __float2int_rz(__fdividef(acosf(dsh_sPhoton[tx].dz), (PI / det_dc[0].na)))*det_dc[0].nr + min(__float2int_rz(__fdividef(__fsqrt_rz(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1);
					AtomicAddULL(&DeviceMem.Tt_ra[index], dsh_sPhoton[tx].weight);
					
					DeviceMem.check[begin + tx].c = 4.0;
					DeviceMem.check[begin + tx].w = dsh_sPhoton[tx].weight;
					dsh_sPhoton[tx].weight = 0;
				}
			}
		}
		//w=0;
		if (dsh_sPhoton[tx].s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered) ���q�̎��ʌ���
			w_temp = __float2uint_rn(layers_dc[dsh_sPhoton[tx].layer].mua*layers_dc[dsh_sPhoton[tx].layer].mutr*__uint2float_rn(dsh_sPhoton[tx].weight));
			dsh_sPhoton[tx].weight -= w_temp;
			DeviceMem.check[begin + tx].c = 5.0;
			if (ignoreAdetection == 0) // Evaluated at compiletime!
			{
				index = (min(__float2int_rz(__fdividef(dsh_sPhoton[tx].z, det_dc[0].dz)), (int)det_dc[0].nz - 1)*det_dc[0].nr + min(__float2int_rz(__fdividef(sqrtf(dsh_sPhoton[tx].x*dsh_sPhoton[tx].x + dsh_sPhoton[tx].y*dsh_sPhoton[tx].y), det_dc[0].dr)), (int)det_dc[0].nr - 1));
				if (index == index_old)
				{
					w += w_temp;
					//sharedp.weight -= __float2uint_rn(layers_dc[sharedp.layer].mua*layers_dc[sharedp.layer].mutr*__uint2float_rn(sharedp.weight)); 
				}
				else// if(w!=0)
				{
					AtomicAddULL(&DeviceMem.A_rz[index_old], w);
					index_old = index;
					w = w_temp;
				}

			}
			// �p�x�v�Z
			Spin(&dsh_sPhoton[tx], &x, &a, layers_dc[dsh_sPhoton[tx].layer].g);
		}



		if (!PhotonSurvive(&dsh_sPhoton[tx], &x, &a) == 1u) // Check if photons survives or not
		{
			DeviceMem.thread_active[begin + tx] = 1;
			LaunchPhoton(&dsh_sPhoton[tx]);
			break;
		}
	}//end main for loop!
	if (ignoreAdetection == 1 && w != 0)
		AtomicAddULL(&DeviceMem.A_rz[index_old], w);

	__syncthreads();//necessary?
	
	DeviceMem.x[begin + tx] = x; //this one also seems to be coherent
	DeviceMem.p[begin + tx] = dsh_sPhoton[tx]; //this one also seems to be coherent


}//end MCd



__device__  void LaunchPhoton(PhotonStruct* p)
{

	p->dead = 0;
	p->layer = 1;
	p->s = 0;
	p->sleft = 0;

	p->x = 0.0f;
	p->y = 0.0f;
	p->z = 0.0f;
	p->dx = 0.0f;
	p->dy = 0.0f;
	p->dz = 1.0f;
	p->rr = 0.0f;
	p->rc = 0.0f;
	p->layer = 1;
	p->weight = *start_weight_dc; //specular reflection!

}

__global__ void LaunchPhoton_Global(MemStruct DevMem)
{
	PhotonStruct* pd = DevMem.p;
	unsigned int* thread_active = DevMem.thread_active;
	unsigned int* num_terminated_photons=DevMem.num_terminated_photons;
	unsigned long long PosData = blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x;
	//First element processed by the block
	if (PosData < num_photons_dc[0]){
	
		//	 LaunchPhoton(&pd[PosData], d_x[PosData], d_a[PosData]);
		pd[PosData].dx = 0.0f;
		pd[PosData].dy = 0.0f;
		pd[PosData].dz = 1.0f;
		pd[PosData].x	= 0.0f;
		pd[PosData].y	= 0.0f;
		pd[PosData].z	= 0.0;
		pd[PosData].layer	= 1;
		pd[PosData].Index = PosData;
		pd[PosData].weight	= (unsigned int)*start_weight_dc;
		thread_active[PosData] = 0;
		num_terminated_photons[0] = 0;

		//DeviceMem->p[begin + tx] = p;//incoherent!?
		
	}
	return;
}
__global__ void LaunchOutput_Global(MemStruct  mem)
{
	unsigned long long PosData = blockIdx.x*blockDim.x + threadIdx.x;
	PosData %= mem.sim->GetRaSize();
	//First element processed by the block
	

		//	 LaunchPhoton(&pd[PosData], d_x[PosData], d_a[PosData]);
		(*mem.Out_Ptr->L) = 0.0;
		(*mem.Out_Ptr->OPL) = 0.0;
		(*mem.Out_Ptr->opl)= 0.0f;
		mem.Out_Ptr->P = 0.0f;
		mem.Out_Ptr->p1 = 0.0f;
		mem.Out_Ptr->Rd_p[PosData]= 0.0;
		mem.Out_Ptr->Rd_ra[PosData] = 0.0;
		mem.Out_Ptr->Rsp= 0.0;
		

		//DeviceMem->p[begin + tx] = p;//incoherent!?

	
	return;
}
__global__ void SetRandpram(hiprandState* devState){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/* Each thread gets same seed, a different sequence number,
	no offset */
	hiprand_init(*dc_Seed, id, 0, &devState[id]);
}

__global__ void InitRng(MemStruct devMem,hiprandState* RndMakerglobal){
	hiprandState RndMakerLocal;
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	RndMakerLocal = RndMakerglobal[id];
	unsigned long long* X = devMem.x;
	unsigned int* A = devMem.a;
	unsigned long long un64PosData = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int TmpRndH, TmpRndL;
	TmpRndH = hiprand(&RndMakerLocal);
	__syncthreads();
	TmpRndL = hiprand(&RndMakerLocal);
	unsigned long long TmpRAXH = ((unsigned long long)TmpRndH) << 32;
	X[un64PosData] = TmpRAXH | (unsigned long long)TmpRndL;
	__syncthreads();
	TmpRndL = hiprand(&RndMakerLocal);
	A[un64PosData] = TmpRndL;
	un64PosData += NUM_DIV_MAKE_RAND*NUM_THREADS_PER_BLOCK_MAKE_RAND;


		

	return;
}
__device__ double SpinTheta(unsigned long long int* x, unsigned int *a, double g){
	double cost;

	if (g == 0.0)
		cost = 2 * rand_MWC_co(x, a) - 1;
	else {
		double temp = (1 - g*g) / (1 - g + 2 * g*rand_MWC_co(x, a));
		cost = (1 + g*g - temp*temp) / (2 * g);
		if (cost < -1) cost = -1;
		else if (cost > 1) cost = 1;
	}
	return(cost);
}
__device__ void Hop(PhotonStruct* p,float s){
	p->x = s*p->dx;
	p->y = s*p->dy;
	p->z = s*p->dz;
}
__device__ void Spin(PhotonStruct* p, unsigned long long int* x, unsigned int *a, float g)
{
	float cost, sint;	// cosine and sine of the 
	// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
	// azimuthal angle psi. 
	float temp=2.1;

	float tempdir = p->dx;
	// Open CUDA Code
	//This is more efficient for g!=0 but of course less efficient for g==0
	//	temp = __fdividef((1.0f - (g)*(g)), (1.0f - (g)+2.0f*(g)*rand_MWC_co(x, a)));//Should be close close????!!!!!
	//	cost = __fdividef((1.0f + (g)*(g)-temp*temp), (2.0f*(g)));
	//	if (g == 0.0f)
	//		cost = 2.0f*rand_MWC_co(x, a) - 1.0f;//Should be close close??!!!!!

	// MIYAHIRA mcml SpinTheta()
	cost = SpinTheta(x,a,g);

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rand_MWC_co(x, a), &sinp, &cosp);// spin psi [0-2*PI)

	temp = sqrtf(1.0f - p->dz*p->dz);

	if (temp == 0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost, p->dz*cost);	// copysign(a,b)==  a*SIGN(b) 
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp), temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp), temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp = rsqrtf(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;

}// end Spin
__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, unsigned long long* x, unsigned int* a, unsigned int* returnvalue)
{
	
	//returnvalue�͐V�K�ǉ��D�߂�l��Ԃ����߂ɐݒu�i16/12/20)
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)
	
	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);
	//refraction index matching automatic transmission and no direction change
	if (n1 == n2)
	{
		*returnvalue = 0;
		p->layer = new_layer;
		//return 0u;
		return 0u;
	}
	//total internal reflection, no layer change but z-direction mirroring
	if ((n1>n2) && ((n2/n1)<sqrtf( 1- (cos_angle_i*cos_angle_i))))
	{
		*returnvalue = 1;
		p->rc = 2;
		p->dz *= -1.0f; 
		
		return 1u;
	}
	//normal incident
	if (cos_angle_i == 1.0f)
	{
		r = __fdividef((n1 - n2), (n1 + n2));
		p->rr = r;
		p->rc = 3;
		if (rand_MWC_co(x, a) <= r*r)
		{
			//reflection, no layer change but z-direction mirroring
			*returnvalue = 1;
			p->dz *= -1.0f;
			
			return 1u;
		}
		else
		{	
			*returnvalue = 0;
			p->rc = 4;
			//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
			//return 0u;
			
		}
	}
	else
	{
		//long and boring calculations of r
		float sinangle_i = sqrtf(1.0f - cos_angle_i*cos_angle_i);
		float sinangle_e = n1/n2*sinangle_i;
		float cosangle_e = sqrtf(1.0f - sinangle_e*sinangle_e);
		
		float cossumangle = (cos_angle_i*cosangle_e) - sinangle_i*sinangle_e;
		float cosdiffangle = (cos_angle_i*cosangle_e) + sinangle_i*sinangle_e;
		float sinsumangle = sinangle_i*cosangle_e + (cos_angle_i*sinangle_e);
		float sindiffangle = sinangle_i*cosangle_e - (cos_angle_i*sinangle_e);
		
		r = 0.5*sindiffangle*sindiffangle*__fdividef((cosdiffangle*cosdiffangle + cossumangle*cossumangle), (sinsumangle*sinsumangle*cosdiffangle*cosdiffangle));
		//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
		// save a few multiplications, calculate cos_angle_i^2;
		//float e = __fdividef(n1*n1, n2*n2)*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		//r = 2 * sqrtf((1.0f - cos_angle_i*cos_angle_i)*(1.0f - e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
		//e = e + (cos_angle_i*cos_angle_i)*(1.0f - 2.0f*e);//Update the value of e
		//r = e*__fdividef((1.0f - e - r), ((1.0f - e + r)*(e + r)));//Calculate r	
		p->rr = r;

	}
	
	
	//p->sleft = p->dz;

	if (rand_MWC_co(x, a) <= r)
	{
		*returnvalue = 1;
		p->rc = 5;
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		
		return 1u;
	}
	else
	{
		*returnvalue = 0;
		p->rc = 6;
		// Transmission, update layer and direction
		r = __fdividef(n1, n2);
		
		float e = r*r*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1 - e), p->dz);
		p->layer = new_layer;
		return 0u;
		
	}
}
__device__ unsigned int PhotonSurvive(PhotonStruct* p, unsigned long long* x, unsigned int* a)
{	//Calculate wether the photon survives (returns 1) or dies (returns 0)

	if (p->weight>WEIGHTI) return 1u; // No roulette needed
	if (p->weight == 0u) return 0u;	// Photon has exited slab, i.e. kill the photon

	if (rand_MWC_co(x, a)<CHANCE)
	{
		p->weight = __float2uint_rn(__fdividef((float)p->weight, CHANCE));
		return 1u;
	}

	//else
	return 0u;
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned long long add)
{
	if (atomicAdd((unsigned long long*)address, add) + add<add)
		atomicAdd(((unsigned long long*)address) + 1, 1u);
}
__device__ void AtomicAddDBL(double* address, double add)
{
	if (atomicAdd((double*)address, add) + add<add)
		atomicAdd(((double*)address) + 1, 1u);
}
#define MBIG 1000000000
#define MSEED 161803398
#define MZ 0
#define FAC 1.0E-9

__device__ float ran3(int *idum)
{
	static int inext, inextp;
	static long ma[56];
	static int iff = 0;
	long mj, mk;
	int i, ii, k;

	if (*idum < 0 || iff == 0) {
		iff = 1;
		mj = MSEED - (*idum < 0 ? -*idum : *idum);
		mj %= MBIG;
		ma[55] = mj;
		mk = 1;
		for (i = 1; i <= 54; i++) {
			ii = (21 * i) % 55;
			ma[ii] = mk;
			mk = mj - mk;
			if (mk < MZ) mk += MBIG;
			mj = ma[ii];
		}
		for (k = 1; k <= 4; k++)
			for (i = 1; i <= 55; i++) {
				ma[i] -= ma[1 + (i + 30) % 55];
				if (ma[i] < MZ) ma[i] += MBIG;
			}
		inext = 0;
		inextp = 31;
		*idum = 1;
	}
	if (++inext == 56) inext = 1;
	if (++inextp == 56) inextp = 1;
	mj = ma[inext] - ma[inextp];
	if (mj < MZ) mj += MBIG;
	ma[inext] = mj;
	return mj*FAC;
}

#undef MBIG
#undef MSEED
#undef MZ
#undef FAC


/***********************************************************
*	Generate a random number between 0 and 1.  Take a
*	number as seed the first time entering the function.
*	The seed is limited to 1<<15.
*	We found that when idum is too large, ran3 may return
*	numbers beyond 0 and 1.
****/
__device__ double RandomNum(void)
{
	static Boolean first_time = 1;
	static int idum;	/* seed for ran3. */

	if (first_time) {
#if STANDARDTEST /* Use fixed seed to test the program. */
		idum = -1;
#else
		idum = -(int)1 % (1 << 15);
		/* use 16-bit integer as the seed. */
#endif
		ran3(&idum);
		first_time = 0;
		idum = 1;
	}

	return(ran3(&idum));
}

__device__ float rand_MWC_co(unsigned long long* x, unsigned int* a)
{
	return RandomNum();
	float temp = 0.0;
	
	//Generate a random number [0,1)
	//*x = (*x & 0xffffffffull)*(*a) + (*x >> 32);
	//temp = __fdividef(__uint2float_rz((unsigned int)(*x)), (float)0x100000000);// The typecast will truncate the x so that it is 0<=x<(2^32-1),__uint2float_rz ensures a round towards zero since 32-bit floating point cannot represent all integers that large. Dividing by 2^32 will hence yield [0,1)
	//
	//return temp;
}//end __device__ rand_MWC_co
__device__ float rand_MWC_oc(unsigned long long* x, unsigned int* a)
{
	//Generate a random number (0,1]
	return 1.0f - rand_MWC_co(x, a);
}//end __device__ rand_MWC_oc


cCUDAMCML::cCUDAMCML(){

}
cCUDAMCML::~cCUDAMCML(){
}
//void cCUDAMCML::RunOldCarnel(){
//	dim3 dimGrid(NUM_GRID_PER_BLOCK);
//	dim3 dimBlock(NUM_THREADS_PER_BLOCK);
//	unsigned int threads_active_total = 1;
//	int i = 0;
//	while (*m_sHostMem.num_terminated_photons < m_simulations->number_of_photons)
//	{
//		i++;
//		//run the kernel
//		if (m_simulations->ignoreAdetection == 1){
//			MCd<1> << <dimGrid, dimBlock >> >(m_sDeviceMem);
//		}
//		else{
//			MCd<0> << <dimGrid, dimBlock >> >(m_sDeviceMem);
//		}
//		hipDeviceSynchronize(); // Wait for all threads to finish
//		hipError_t cudastat = hipGetLastError(); // Check if there was an error
//
//		// Copy thread_active from device to host
//		hipMemcpy(m_sHostMem.thread_active, m_sDeviceMem.thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyDeviceToHost);
//		threads_active_total = 0;
//		for (int ii = 0; ii < NUM_THREADS; ii++){
//			threads_active_total += m_sHostMem.thread_active[ii];
//		}
//
//		hipMemcpy(m_sHostMem.num_terminated_photons, m_sDeviceMem.num_terminated_photons, sizeof(unsigned int), hipMemcpyDeviceToHost);
//		hipMemcpy(m_sHostMem.p, m_sDeviceMem.p, sizeof(PhotonStruct)*m_simulations->number_of_photons, hipMemcpyDeviceToHost);
//	}
//	CopyDeviceToHostMem(&m_sHostMem, &m_sDeviceMem, m_simulations);
//}
// �v�Z�̒���
int cCUDAMCML::MakeRandTableDev(){
	hiprandState *devStates;
	hipError_t  cudastat;
	dim3 dimNumBlockRand(NUM_GRID_PER_BLOCK);
	dim3 dimNumThreadRand(NUM_THREADS_PER_BLOCK);
	hipMalloc((void **)&devStates, NUM_THREADS * sizeof(hiprandState));
	// �V�[�h�C�����l�Ƃ��ėp���闐���z��̍쐬
	// MCML�̗��������ɗ��p�ł��Ȃ��@�ˁ@���C�u�����̓s����C�X���b�h������������邽��
	SetRandpram << < dimNumBlockRand, dimNumThreadRand >> > (devStates);
	cudastat = hipGetLastError();	// Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_RND_;
	}
	hipDeviceSynchronize();
	InitRng << < dimNumBlockRand, dimNumThreadRand >> > (m_sDeviceMem,devStates);
	// ���ؗp
	hipMemcpy(m_sHostMem.a, m_sDeviceMem.a, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(m_sHostMem.x, m_sDeviceMem.x, NUM_THREADS * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	cudastat = hipGetLastError();	// Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_RND_;
	}
	return 0;
}

int cCUDAMCML::InitOutput(){


	dim3 dimNumBlock(19);
	dim3 dimNumThread(NUM_THREADS_PER_BLOCK);

	LaunchOutput_Global << < dimNumBlock, dimNumThread >> > (m_sDeviceMem);
	hipError_t cudastat = hipGetLastError();	// Check if there was an error

	if (cudastat){
		return _ERR_GPU_SIM_LANCH_PHOTON_;
	}
	return 0;
}
int cCUDAMCML::InitPhoton(){


	dim3 dimNumBlock(19);
	dim3 dimNumThread(NUM_THREADS_PER_BLOCK);

	LaunchPhoton_Global << < dimNumBlock, dimNumThread >> > (m_sDeviceMem);
	hipError_t cudastat = hipGetLastError();	// Check if there was an error

	if (cudastat){
		return _ERR_GPU_SIM_LANCH_PHOTON_;
	}
	return 0;
}
int cCUDAMCML::DoOneSimulation(SimulationStruct* simulation)
{

	unsigned int threads_active_total = simulation->number_of_photons;
	unsigned int i;

	hipError_t cudastat;

	int STAT = 0;
	// Start the clock

	dim3 dimNumBlock(NUM_GRID_PER_BLOCK);
	dim3 dimNumThread(NUM_THREADS_PER_BLOCK);
	int TotalP = 0;
	//ReflectTest<<<dimNumBlock,dimNumThread>>>(m_sDeviceMem);
	
	while (TotalP<simulation->number_of_photons)
	{
		
		//run the kernel
		if (simulation->ignoreAdetection == 1){
			CalcMCGPU<1> << < dimNumBlock, dimNumThread >> >(m_sDeviceMem);
			
		}
		else{
			CalcMCGPU<0> << < dimNumBlock, dimNumThread >> >(m_sDeviceMem);
	
		}
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat){
			return _ERR_GPU_SIM_MCML_;
		}

	

		// ���ؗp
		cudastat = hipMemcpy(m_sHostMem.p, m_sDeviceMem.p, NUM_THREADS * sizeof(PhotonStruct), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.x, m_sDeviceMem.x, NUM_THREADS * sizeof(unsigned long  long), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.a, m_sDeviceMem.a, NUM_THREADS * sizeof(unsigned int ), hipMemcpyDeviceToHost);
		

		cudastat = hipMemcpy(m_sHostMem.thread_active, m_sDeviceMem.thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.num_terminated_photons, m_sDeviceMem.num_terminated_photons,sizeof(unsigned int), hipMemcpyDeviceToHost);
		cudastat = hipMemcpy(m_sHostMem.check, m_sDeviceMem.check, NUM_THREADS * sizeof(CheckStruct), hipMemcpyDeviceToHost);
		//	hipDeviceSynchronize();		// Wait for all threads to finish
		// 
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat){
			return _ERR_GPU_SIM_MEMCPY_;
		}
		//std::ofstream ofs("text.csv");
		//for (int i = 0; i < NUM_THREADS; i++){
		//	ofs << m_sHostMem.check[i].dz << std::endl;
		//	//ofs << m_sHostMem.p[i].dead << ",";
		//	//ofs << m_sHostMem.p[i].sleft << ",";
		//	//ofs << m_sHostMem.p[i].rr << std::endl;
		//}
		int x = 0;
		for (int i = 0; i < NUM_THREADS; i++){
			if (m_sHostMem.thread_active[i] != 65535){
				TotalP += m_sHostMem.thread_active[i];
			}
			if (m_sHostMem.check[i].c != 0 ){

				x++;

			}
		}
	}

	cudastat = hipGetLastError(); // Check if there was an error
	if (cudastat){
		return _ERR_GPU_SIM_ANOTHER_;
	}


	CopyDeviceToHostMem(&m_sHostMem, &m_sDeviceMem, simulation);
	return _SUCCESS_GPU_SIM_;

}
int cCUDAMCML::InitMallocMem(SimulationStruct* sim){
	unsigned int State = 0;
	hipError_t tmp;

	tmp = hipMalloc((void**)&m_sDeviceMem, sizeof(MemStruct));
	if (tmp != hipSuccess) {
		State |= 0x01;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.p, (NUM_THREADS)*sizeof(PhotonStruct));
	if (tmp != hipSuccess) {
		State |= 0x01;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.x, (NUM_THREADS)*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x02;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.a, (NUM_THREADS)*sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x04;
	}
	
	tmp = hipMalloc((void**)&m_sDeviceMem.thread_active, (NUM_THREADS)*sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.num_terminated_photons, sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x40;
	}
	unsigned int rz_size = sim->GetRzSize(); 
	unsigned int ra_size = sim->GetRaSize(); 
	tmp = hipMalloc((void**)&m_sDeviceMem.A_rz, rz_size *sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x80;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.Rd_ra, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x100;
	}
	tmp = hipMalloc((void**)&m_sDeviceMem.Tt_ra, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x200;
	}

	tmp = hipMalloc((void**)&m_sDeviceMem.sim, sizeof(SimulationStruct));
	if (tmp != hipSuccess) {
		State |= 0x400;
	}

	tmp = hipMalloc((void**)&m_sDeviceMem.check, NUM_THREADS*sizeof(CheckStruct));
	if (tmp != hipSuccess) {
		State |= 0x400;
	}

	tmp = hipMalloc((void**)&m_sDeviceMem.Out_Ptr, sizeof(OutStruct));
	if (tmp != hipSuccess) {
		State |= 0x10;
	}
	/*
	���̎��_�ł�m_sDeviceMem.Out_Ptr�ɂ�device��Ŋm�ۂ����������̈ʒu(�A�h���X)��Host��������ɕۑ�����Ă���
	��: Out_Ptr �� device��������2�Ԗڂ�擪�Ƃ��Ċm�ۂ��C����device��̃A�h���X��Host��������5�Ԗڂɕۑ������ꍇ
					 1  2  3  4  5
	Host�̃�����	[0][0][0][0][2]			
	Dev �̃�����	[0][d      ][0]			[d]�E�E�EOut_Ptr���m�ۂ����ꏊ(data��d) ����Ȋ���
	
	���̏�Ԃ�	tmp = hipMalloc((void**)&m_sDeviceMem.Out_Ptr->Rd_ra, 8);�@�����s����Ƃǂ��Ȃ邩�H
	���������@m_sDeviceMem.Out_Ptr->Rd_ra�@�͂ǂ̂悤�ȈӖ��������Ƃ����Ɓ@m_sDeviceMem.Out_Ptr����ɂ��đ��ΓI��Rd_ra�̃������̈ʒu�������Ă���
	��}�Ō�����[d]�Ŋm�ۂ��Ă���2~4�Ԗڂ�3���̈ʒu���������Ƃ��ł���D

	���������ɗ��Ƃ���������Dm_sDeviceMem.Out_Ptr->Rd_ra�����ۂɋ��߂�̂�CPU�C�܂�Host��������ōl�����Ă��܂�
	���̂��� m_sDeviceMem.Out_Ptr->Rd_ra�@�͏�}�Ō�����Host���������� 2�Ԗڂ��Q�Ƃ��Ă��܂��D���R�����ɂ͉����Ȃ��̂ŃG���[���N���Ă��܂�


	CPU�u������Ƃ܂��āI2�Ԗڂ̃������t�ߊm�ۂ��ĂȂ����I�v�ˁ@�G���[����
						_________
					   ��		 |
					 1  2  3  4  5
	Host�̃�����	[0][0][0][0][2]
	Dev �̃�����	[0][d      ][0]			[d]�E�E�EOut_Ptr���m�ۂ����ꏊ(data��d) ����Ȋ���

	�E�Ώ����@�͍\���̂�Host���ŉ��m�ہ�Device�ɓ]������Ηǂ�
	�Ⴆ�ΓK����Host�̏ꏊ�ɍ\���̂�p�ӂ�
		OutStruct TmpOutStruct;
		tmp = hipMalloc((void**)&TmpOutStruct.Rd_ra, 1);
		tmp = hipMalloc((void**)&TmpOutStruct.L, 1);
		tmp = hipMalloc((void**)&TmpOutStruct.OP, 1);
	���̂悤�ɂ���Ɖ��̐}�̂悤�ɂȂ�D

					 1  2  3  4  5  6   7  8
	Host�̃�����    [6  7  8][0][2]						[6 7 8]	�E�E�ETmpOutStruct�̒��g(device�̏ꏊ)
	Dev �̃�����	[0][d      ][0][Rd][L][OP]			[d]		�E�E�EOut_Ptr���m�ۂ����ꏊ(data��d) ����Ȋ���

	���̂��Ƃ�hipMemcpy(m_sDeviceMem.Out_Ptr, &TmpOutStruct,sizeof(OutStruct), hipMemcpyHostToDevice);���s�����Ƃ�
	m_sDeviceMem.Out_Ptr�̒��g���m�ۂł���D

					 1  2  3  4  5  6   7  8
	Host�̃�����    [6]  7  8][0][2]						[6 7 8]	�E�E�ETmpOutStruct�̒��g(device�̏ꏊ)
	Dev �̃�����	[0][6  7  8][0][Rd][L][OP]			[d]		�E�E�EOut_Ptr���m�ۂ����ꏊ(data��d) ����Ȋ���

	�\�[�X��2ch(http://toro.2ch.net/test/read.cgi/tech/1314104886/)[108~111�Q��]
	*/
	
	tmp = hipMalloc((void**)&m_sOutStruct.Rd_ra, ra_size*sizeof(double));

	
	tmp = hipMalloc((void**)&m_sOutStruct.Rd_p, ra_size*sizeof(double));
//	if (tmp != hipSuccess) {
//		State |= 0x800;
//	}
	tmp = hipMalloc((void**)&m_sOutStruct.P, sizeof(double));
//	if (tmp != hipSuccess) {
//		State |= 0x1000;
//	}
	tmp = hipMalloc((void**)&m_sOutStruct.p1, sizeof(double));
	//	if (tmp != hipSuccess) {
//		State |= 0x2000;
//	}
	tmp = hipMalloc((void**)&m_sOutStruct.Rsp, sizeof(double));
	//	if (tmp != hipSuccess) {
//		State |= 0x4000;
//	}
	tmp = hipMalloc((void**)&m_sOutStruct.L, sizeof(double));
	//	if (tmp != hipSuccess) {
//		State |= 0x8000;
//	}
	tmp = hipMalloc((void**)&m_sOutStruct.OPL, sizeof(double));
	
	
	//	if (tmp != hipSuccess) {
//		State |= 0x10000;
//	}
	tmp = hipMalloc((void**)&m_sOutStruct.opl, sizeof(double));
//	if (tmp != hipSuccess) {
//		State |= 0x20000;
//	}
	
	hipMemcpy(m_sDeviceMem.Out_Ptr, &m_sOutStruct, sizeof(OutStruct), hipMemcpyHostToDevice);
	
	
	

	// Allocate p on the device!!
	// Allocate A_rz on host and device
	m_sHostMem.p = new PhotonStruct			[NUM_THREADS];
	m_sHostMem.x = new unsigned long long	[NUM_THREADS];
	m_sHostMem.a = new unsigned int			[NUM_THREADS];
	
	if ((m_sHostMem.x == NULL) || (m_sHostMem.a == NULL)){
		State |= 0x00008000;
	}
	m_sHostMem.A_rz = new unsigned long long [rz_size];
	if (m_sHostMem.A_rz == NULL){
		State |= 0x00010000;
	}
	m_sHostMem.Rd_ra = new unsigned long long[ra_size];
	if (m_sHostMem.Rd_ra == NULL){
		State |= 0x00020000;
	}
	m_sHostMem.Tt_ra = new unsigned long long[ra_size];
	if (m_sHostMem.Tt_ra == NULL){
		State |= 0x00040000;
	}
	// Allocate thread_active on the device and host
	m_sHostMem.thread_active = new unsigned int[sim->number_of_photons];
	if (m_sHostMem.thread_active == NULL){
		State |= 0x00080000;
	}

	m_sHostMem.num_terminated_photons = new unsigned int[1];
	if (m_sHostMem.num_terminated_photons == NULL){
		State |= 0x00100000;
	}
	*m_sHostMem.num_terminated_photons = 0;

	m_sHostMem.Out_Ptr = new OutStruct;
	if (m_sHostMem.Out_Ptr == NULL){
		State |= 0x00200000;
	}

	m_sHostMem.Out_Ptr->Rd_ra = new double[ra_size];
	if (m_sHostMem.Out_Ptr->Rd_ra == NULL){
		State |= 0x00400000;
	}

	m_sHostMem.Out_Ptr->Rd_p = new double[ra_size];
	if (m_sHostMem.Out_Ptr->Rd_p == NULL){
		State |= 0x00800000;
	}
	m_sHostMem.Out_Ptr->OPL = new double, sizeof(double);
	if (m_sHostMem.Out_Ptr->OPL == NULL){
		State |= 0x001000000;
	}
	m_sHostMem.Out_Ptr->L = new double ,sizeof(double);
	if (m_sHostMem.Out_Ptr->L == NULL){
		State |= 0x0020000000;
	}
	m_sHostMem.Out_Ptr->opl = new double ,sizeof (double);
	if (m_sHostMem.Out_Ptr->opl == NULL){
		State |= 0x0040000000;
	}
	m_sHostMem.sim = new SimulationStruct, sizeof(SimulationStruct);
	if (m_sHostMem.sim == NULL){
		State |= 0x0080000000;
	}
	m_sHostMem.check = new CheckStruct[ NUM_THREADS];
	if (m_sHostMem.check == NULL){
		State |= 0x0100000000;
	}

	return State ;
}
void cCUDAMCML::CopyDeviceToHostMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{ //Copy data from Device to Host memory

	int rz_size = sim->det.nr*sim->det.nz;
	int ra_size = sim->det.nr*sim->det.na;
	hipError_t tmp;
	// Copy A_rz, Rd_ra and Tt_ra
	tmp=hipMemcpy(HostMem->A_rz, DeviceMem->A_rz, rz_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);
	tmp=hipMemcpy(HostMem->Rd_ra, DeviceMem->Rd_ra, ra_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);
	tmp=hipMemcpy(HostMem->Tt_ra, DeviceMem->Tt_ra, ra_size*sizeof(unsigned long long), hipMemcpyDeviceToHost);
	tmp=hipMemcpy(HostMem->check, DeviceMem->check, NUM_THREADS*sizeof(CheckStruct), hipMemcpyDeviceToHost);
	tmp=hipMemcpy(HostMem->Out_Ptr->Rd_ra,m_sOutStruct.Rd_ra, ra_size*sizeof(double), hipMemcpyDeviceToHost);
	tmp=hipMemcpy(HostMem->Out_Ptr->Rd_p, m_sOutStruct.Rd_p, ra_size*sizeof(double), hipMemcpyDeviceToHost);
	
	//Also copy the state of the RNG's
	hipMemcpy(HostMem->p, DeviceMem->p, NUM_THREADS *sizeof(PhotonStruct), hipMemcpyDeviceToHost);
	tmp = hipMemcpy(HostMem->Out_Ptr->opl, m_sOutStruct.opl, sizeof(double), hipMemcpyDeviceToHost);
	
	if (tmp != hipSuccess) {

	}
	return ;
}
int cCUDAMCML::CopyHostToDeviceMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim){
	// Allocate x and a on the device (For MWC RNG)
	hipError_t tmp;
	int State = 0;
	tmp = hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS*sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x40;
	}


	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x80;
	}
	
	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x200;
	}
	tmp = hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x800;
	}
	tmp = hipMemcpy(DeviceMem->Out_Ptr->OPL, HostMem->Out_Ptr->OPL, sizeof(double), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x1000;
	}
	tmp = hipMemcpy(DeviceMem->Out_Ptr->opl, HostMem->Out_Ptr->opl, sizeof(double), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x2000;
	}
	tmp = hipMemcpy(DeviceMem->Out_Ptr->L, HostMem->Out_Ptr->L, sizeof(double), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x4000;
	}
	return State;
}
int cCUDAMCML::InitDCMem(SimulationStruct* sim)
{
	hipError_t tmp;
	int State = 0;
	// Copy det-data to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(det_dc), &(sim->det), sizeof(DetStruct));
	if (tmp != hipSuccess) {
		State |= 0x1;
	}

	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc), &(sim->n_layers), sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x2;
	}

	// Copy start_weight_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc), &(sim->start_weight), sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x4;
	}

	// Copy layer data to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(layers_dc), sim->layers, (sim->n_layers + 2)*sizeof(LayerStruct));
	if (tmp != hipSuccess) {
		State |= 0x8;
	}

	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc), &(sim->number_of_photons), sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x10;
	}
	// Copy num_photons_dc to constant device memory
	tmp = hipMemcpyToSymbol(HIP_SYMBOL(dc_Seed), &(sim->Seed), sizeof(unsigned int));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}
	


	return State;

}
int cCUDAMCML::InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{
	int rz_size, ra_size;
	hipError_t tmp;
	rz_size = sim->det.nr*sim->det.nz;
	ra_size = sim->det.nr*sim->det.na;
	int Total = 0;

	// Allocate p on the device!!
	hipMalloc((void**)&DeviceMem->p, NUM_THREADS*sizeof(PhotonStruct));
	Total += NUM_THREADS*sizeof(PhotonStruct);
	// Allocate A_rz on host and device
	HostMem->A_rz = new(unsigned long long)(rz_size*sizeof(unsigned long long));
	if (HostMem->A_rz == NULL){ 
		printf("Error allocating HostMem->A_rz"); 
		exit(1); 
	}
	hipMalloc((void**)&DeviceMem->A_rz, rz_size*sizeof(unsigned long long));
	Total += rz_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->A_rz, 0, rz_size*sizeof(unsigned long long));

	// Allocate Rd_ra on host and device
	HostMem->Rd_ra = new(unsigned long long)(ra_size*sizeof(unsigned long long));
	if (HostMem->Rd_ra == NULL){ 
		printf("Error allocating HostMem->Rd_ra"); 
		exit(1);
	}
	hipMalloc((void**)&DeviceMem->Rd_ra, ra_size*sizeof(unsigned long long));
	Total += ra_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->Rd_ra, 0, ra_size*sizeof(unsigned long long));

	// Allocate Tt_ra on host and device
	HostMem->Tt_ra = new(unsigned long long)(ra_size*sizeof(unsigned long long));
	if (HostMem->Tt_ra == NULL){ printf("Error allocating HostMem->Tt_ra"); exit(1); }
	hipMalloc((void**)&DeviceMem->Tt_ra, ra_size*sizeof(unsigned long long));
	Total += ra_size*sizeof(unsigned long long);
	hipMemset(DeviceMem->Tt_ra, 0, ra_size*sizeof(unsigned long long));


	// Allocate x and a on the device (For MWC RNG)
	hipMalloc((void**)&DeviceMem->x, NUM_THREADS*sizeof(unsigned long long));
	Total += NUM_THREADS*sizeof(unsigned long long);

	hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS*sizeof(unsigned long long), hipMemcpyHostToDevice);
	tmp = hipMalloc((void**)&DeviceMem->a, NUM_THREADS*sizeof(unsigned int));
	Total += NUM_THREADS*sizeof(unsigned long long);

	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);


	// Allocate thread_active on the device and host
	HostMem->thread_active = new unsigned int (NUM_THREADS*sizeof(unsigned int));
	if (HostMem->thread_active == NULL){ printf("Error allocating HostMem->thread_active"); exit(1); }
	for (int i = 0; i < NUM_THREADS; i++){
		HostMem->thread_active[i] = 1u;
	}

	Total += NUM_THREADS*sizeof(unsigned int);
	tmp = hipMalloc((void**)&(DeviceMem->thread_active), NUM_THREADS*sizeof(unsigned int));

	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);


	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = new(unsigned int)(sizeof(unsigned int));
	if (HostMem->num_terminated_photons == NULL){ printf("Error allocating HostMem->num_terminated_photons"); exit(1); }
	*HostMem->num_terminated_photons = 0;

	hipMalloc((void**)&DeviceMem->num_terminated_photons, sizeof(unsigned int));
	hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);

	return 1;
}
int cCUDAMCML::InitContentsMem(SimulationStruct* sim)
{
	MemStruct* DeviceMem = &m_sDeviceMem;
	MemStruct* HostMem = &m_sHostMem;

	int State=0;
	int rz_size, ra_size;
	hipError_t tmp;
	rz_size = sim->det.nr*sim->det.nz;
	ra_size = sim->det.nr*sim->det.na;

	tmp = hipMemset(DeviceMem->A_rz, 0, rz_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x02;
	}

	tmp = hipMemset(DeviceMem->Rd_ra, 0, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x08;
	}

	tmp = hipMemset(DeviceMem->Tt_ra, 0, ra_size*sizeof(unsigned long long));
	if (tmp != hipSuccess) {
		State |= 0x20;
	}

	PhotonStruct TmpPS;

	HostMem->p->x = 0;
	HostMem->p->y = 0;
	HostMem->p->z = 0;
	HostMem->p->dx = 0;
	HostMem->p->dy = 0;
	HostMem->p->dz = 0;
	HostMem->p->weight = 0;
	HostMem->p->layer = 0;
	HostMem->p->Index = 0;
	HostMem->p->dead = 0;
	HostMem->p->sleft = 0;

	HostMem->p->rr = 0;
	HostMem->p->rc = 0;
	tmp = hipMemset(DeviceMem->p, 0, NUM_THREADS *sizeof(int));
	if (tmp != hipSuccess) {
		State |= 0x40;
	}
	
	for (int i = 0; i < NUM_THREADS; i++){
		HostMem->check[i].c = 0.0;
		HostMem->check[i].w = 0.0;
		HostMem->check[i].cc = 0.0;
		HostMem->check[i].dz = 0.0;
		HostMem->check[i].r = 0.0;
	
	}



	tmp = hipMemcpy(DeviceMem->check, HostMem->check, NUM_THREADS *sizeof(CheckStruct),hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x1000;
	}
	tmp = hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS *sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x80;
	}


	tmp = hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x100;
	}

	for (int i = 0; i < sim->number_of_photons; i++){
		HostMem->thread_active[i] = 1u;
	}



	tmp = hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x200;
	}



	tmp = hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x400;
	}

	HostMem->sim = m_simulations;
	tmp = hipMemcpy(DeviceMem->sim, HostMem->sim, sizeof(SimulationStruct), hipMemcpyHostToDevice);
	if (tmp != hipSuccess) {
		State |= 0x800;
	}
	return State;
}

void cCUDAMCML::FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{

	hipFree(DeviceMem->p);
	hipFree(DeviceMem->x);
	hipFree(DeviceMem->a);
	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);
	hipFree(DeviceMem->A_rz);
	hipFree(DeviceMem->Rd_ra);
	hipFree(DeviceMem->Tt_ra);
	hipFree(m_sOutStruct.Rd_p);
	hipFree(m_sOutStruct.Rd_ra);
	hipFree(m_sOutStruct.L);
	hipFree(m_sOutStruct.OPL);
	hipFree(m_sOutStruct.opl);
	hipFree(DeviceMem->check);
	

	delete[] HostMem->p;
	delete[] HostMem->x;
	delete[] HostMem->a;
	delete[] HostMem->thread_active;
	delete[] HostMem->num_terminated_photons;
	delete[] HostMem->Reserve;
	delete[] HostMem->A_rz;
	delete[] HostMem->Rd_ra;
	delete[] HostMem->Tt_ra;
	delete[] HostMem->Out_Ptr->Rd_ra;
	delete[] HostMem->Out_Ptr->Rd_p;
	delete[] HostMem->Out_Ptr->L;
	delete[] HostMem->Out_Ptr->OPL;
	delete[] HostMem->Out_Ptr->opl;
	delete[] HostMem->check;
}

void cCUDAMCML::FreeSimulationStruct(SimulationStruct* sim, int nRun)
{
	FreeMemStructs(&m_sHostMem, &m_sDeviceMem);
	for (int i = 0; i < nRun; i++){
		delete sim[i].layers;
	}
	delete[] sim;
	//hipDeviceReset();
}

void cCUDAMCML::FreeFailedSimStrct(SimulationStruct* Sim, int nRun)
{
	for (int i = 0; i < nRun; i++){
		delete Sim[i].layers;
	}
	delete[] Sim;

}


bool cCUDAMCML::CheckGPU(){
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0){
		return 0;
	}
	hipSetDevice(0);
	hipGetDeviceProperties(&m_sDevProp, 0);
	return 1;
}
void cCUDAMCML::InitGPUStat(){
	m_ProcessTime = 0;
	m_un64Membyte = 0;
	m_un64NumPhoton = 0;
	m_un64PrcsDataNum = 0;
	hipDeviceReset();
}
/***********************************************************
*	Record the photon weight exiting the first layer(uz<0),
*	no matter whether the layer is glass or not, to the
*	reflection array.
*
*	Update the photon weight as well.
****/
extern "C"{
	__device__ void RemodelRecordR(MemStruct  DeviceMem, PhotonStruct *p)
	{
		SimulationStruct*sim = DeviceMem.sim;
		OutStruct	*Out_Ptr = DeviceMem.Out_Ptr;
		double	Refl = p->rr;
		double x = p->x;
		double y = p->y;
		double dx = p->dx;
		double dy = p->dy;
		double dz = p->dz;
		double t;
		double r;
		double r1 = sim->r;
		double t1, t2, t3;
		short  it, ia;	/* index to r & angle. */
		double itd, iad;	/* LW 5/20/98. To avoid out of short range.*/
		short  nl = sim->n_layers;
		short	 l;
		int	 n = Out_Ptr->p1;
		int id;
		//Block index
		int bx = blockIdx.x;

		//Thread index
		int tx = threadIdx.x;


		//First element processed by the block
		int begin = blockDim.x*bx;
		
		r = sqrt(x*x + y*y);
		//Out_Ptr->Rd_ra[tx] = tx;
		if (r >= r1 && r <= (r1 + 0.1*r1))
			
		{
			if (y >= 0)
				t1 = atan2(y, x) * 180 / PI;
			else
				t1 = 360 + atan2(y, x) * 180 / PI;

			if (dy <= 0)
				t2 = atan2(-dy, -dx) * 180 / PI;
			else
				t2 = 360 + atan2(-dy, -dx) * 180 / PI;

			t3 = t2 - t1;
			if (t3 < 0)
				t = 360 + t3;
			else
				t = t3;

			itd = (short)(t / sim->dt);
			if (itd > sim->nr - 1) it = sim->nr - 1;
			else it = itd;

			iad = (short)(acos(-dz) * 180 / PI / sim->da);
			if (iad > sim->na - 1) ia = sim->na - 1;
			else ia = iad;
			//AtomicAddDBL(&Out_Ptr->Rd_ra[sim->nr*it + ia], p->weight*(1.0 - Refl));

			AtomicAddDBL(&Out_Ptr->Rd_ra[sim->nr*ia + it], p->weight*(1.0 - Refl));
			//Out_Ptr->Rd_ra[tx] = tx;		/* �e�V���p�E�e���ʊp�̌��q�E�F�C�g�̋L�^ */
			//AtomicAddDBL(&Out_Ptr->Rd_p[sim->nr*it + ia] ,1);							/* �e�V���p�E�e���ʊp�̌��q���̋L�^ */
			AtomicAddDBL(&Out_Ptr->Rd_p[sim->nr*ia + it], 1);
			//AtomicAddDBL(&Out_Ptr->Rd_p[tx], 1);
			AtomicAddDBL(&Out_Ptr->P , p->weight*(1.0 - Refl));

			for (l = 1; l <= nl; l++)
			{
				Out_Ptr->L[l] += Out_Ptr->OPL[l] * p->weight*(1.0 - Refl);		/* ����G���A�ɓ��������q�̌��H���̋L�^ */

				Out_Ptr->p1 += 1;
			}
		}

		p->weight *= Refl;
	}
	//__host__ __device__ void RecordR(double			Refl,	/* reflectance. */
	//	InputStruct  *	In_Ptr,
	//	PhotonStruct *	p,
	//	OutStruct *	Out_Ptr)
	//{
	//	double x = p->x;
	//	double y = p->y;
	//	double dx = p->dx;
	//	double dy = p->dy;
	//	double dz = p->dz;
	//	double t;
	//	double r;
	//	double r1 = In_Ptr->r;
	//	double t1, t2, t3;
	//	short  it, ia;	/* index to r & angle. */
	//	double itd, iad;	/* LW 5/20/98. To avoid out of short range.*/
	//	short  nl = In_Ptr->num_layers;
	//	short	 l;
	//	int	 n = Out_Ptr->p1;
	//
	//	r = sqrt(x*x + y*y);
	//
	//	if (r >= r1 && r <= (r1 + 0.1*r1))
	//	{
	//		if (y >= 0)
	//			t1 = atan2(y, x) * 180 / PI;
	//		else
	//			t1 = 360 + atan2(y, x) * 180 / PI;
	//
	//		if (dy <= 0)
	//			t2 = atan2(-dy, -dx) * 180 / PI;
	//		else
	//			t2 = 360 + atan2(-dy, -dx) * 180 / PI;
	//
	//		t3 = t2 - t1;
	//		if (t3<0)
	//			t = 360 + t3;
	//		else
	//			t = t3;
	//
	//		itd = (short)(t / In_Ptr->dt);
	//		if (itd>In_Ptr->nr - 1) it = In_Ptr->nr - 1;
	//		else it = itd;
	//
	//		iad = (short)(acos(-dz) * 180 / PI / In_Ptr->da);
	//		if (iad>In_Ptr->na - 1) ia = In_Ptr->na - 1;
	//		else ia = iad;
	//
	//		Out_Ptr->Rd_ra[it][ia] += p->weight*(1.0 - Refl);		/* �e�V���p�E�e���ʊp�̌��q�E�F�C�g�̋L�^ */
	//		Out_Ptr->Rd_p[it][ia] += 1;							/* �e�V���p�E�e���ʊp�̌��q���̋L�^ */
	//		Out_Ptr->P += p->weight*(1.0 - Refl);
	//
	//		for (l = 1; l <= nl; l++)
	//			Out_Ptr->L[l] += Out_Ptr->OPL[l] * p->weight*(1.0 - Refl);		/* ����G���A�ɓ��������q�̌��H���̋L�^ */
	//
	//		Out_Ptr->p1 += 1;
	//	}
	//
	//	p->weight *= Refl;
	//}
	__host__ __device__ void InitOutputData(MemStruct deviceMem, SimulationStruct sim,
		OutStruct * Out_Ptr)
	{
		short nr = sim.nr;
		short na = sim.na;
		short nl = sim.n_layers;
		/* remember to use nl+2 because of 2 for ambient. */

		if (nr <= 0 || na <= 0 || nl <= 0)
			//nrerror("Wrong grid parameters.\n");

			/* Init pure numbers. */
			Out_Ptr->Rsp = 0.0;

		/* Allocate the arrays and the matrices. */
		Out_Ptr->Rd_ra = AllocMatrix(0, nr - 1, 0, na - 1);
		Out_Ptr->Rd_p = AllocMatrix(0, nr - 1, 0, na - 1);

		Out_Ptr->OPL = AllocVector(0, nl + 1);
		Out_Ptr->L = AllocVector(0, nl + 1);
		Out_Ptr->opl = AllocVector(0, nl + 1);
		ReportResult(sim,*Out_Ptr);
	}
	__host__ __device__ void ReportResult(SimulationStruct sim, OutStruct Out_Parm)
	{
		char time_report[STR_LEN];

		strcpy(time_report, " Simulation time of this run.");
		PunchTime(1, time_report);

		SumScaleResult(sim, &Out_Parm);
		WriteResult(sim, Out_Parm, time_report);
	}
	__host__ __device__ time_t PunchTime(char F, char *Msg)
	{
#if GNUCC
		return(0);
#else
		static clock_t ut0;	/* user time reference. */
		static time_t  rt0;	/* real time reference. */
		double secs;
		char s[STR_LEN];

		if (F == 0) {
			ut0 = clock();
			rt0 = time(NULL);
			return(0);
		}
		else if (F == 1)  {
			secs = (clock() - ut0) / (double)CLOCKS_PER_SEC;
			if (secs < 0) secs = 0;	/* clock() can overflow. */
			sprintf(s, "User time: %8.0lf sec = %8.2lf hr.  %s\n",
				secs, secs / 3600.0, Msg);
			puts(s);
			strcpy(Msg, s);
			return(difftime(time(NULL), rt0));
		}
		else if (F == 2) return(difftime(time(NULL), rt0));
		else return(0);
#endif
	}
	__host__ __device__ void SumScaleResult(SimulationStruct sim, OutStruct * Out_Ptr)
	{
		CalOPL_SD(sim, Out_Ptr);
	}
	__host__ void WriteResult(SimulationStruct sim,
		OutStruct Out_Parm,
		char * TimeReport)
	{
		FILE *file;

		file = fopen(sim.outp_filename, "w");
		//	if (file == NULL) nrerror("Cannot open file to write.\n");

		if (toupper(sim.out_fformat) == 'A')
			WriteVersion(file, "A1");
		else
			WriteVersion(file, "B1");

		fprintf(file, "# %s", TimeReport);
		fprintf(file, "\n");

		WriteInParm(file, sim);
		/* reflectance, absorption, transmittance. */

		/* 1D arrays. */

		/* 2D arrays. */
		WriteRd_ra(file, sim.nr, sim.na, Out_Parm);
		WriteRd_p(file, sim.nr, sim.na, Out_Parm);
		WriteOPL(file, sim.n_layers, Out_Parm);

		fclose(file);
	}
	__device__ __host__ void CalOPL_SD(SimulationStruct sim, OutStruct * Out_Ptr)
	{
		short l;
		short	nl = sim.n_layers;

		for (l = 1; l <= nl; l++)
			Out_Ptr->opl[l] = Out_Ptr->L[l] / Out_Ptr->P;		/* ��n�w�ɓ��������q�̌��H���̕��� */
	}

	__device__ __host__ double *AllocVector(short nl, short nh)
	{
		double *v;
		short i;

		v = (double *)malloc((unsigned)(nh - nl + 1)*sizeof(double));
		//	if (!v) nrerror("allocation failure in vector()");

		v -= nl;
		for (i = nl; i <= nh; i++) v[i] = 0.0;	/* init. */
		return v;
	}
	__host__ void WriteVersion(FILE *file, char *Version)
	{
		fprintf(file,
			"%s \t# Version number of the file format.\n\n",
			Version);
		fprintf(file, "####\n# Data categories include: \n");
		fprintf(file, "# InParm, RAT, \n");
		fprintf(file, "# Rd_ta, \n####\n\n");
	}
	/***********************************************************
	*	Write the input parameters to the file.
	****/
	void WriteInParm(FILE *file, SimulationStruct sim)
	{
		short i;
	
		fprintf(file,
			"InParm \t\t\t# Input parameters. cm is used.\n");
    
		fprintf(file,
			"%s \tA\t\t# output file name, ASCII.\n",
			sim.outp_filename);
		fprintf(file,
			"%ld \t\t\t# No. of photons\n", sim.number_of_photons);
		fprintf(file,
			"%.2lf \t\t\t# No. of SD distance\n", sim.r);

		fprintf(file,
			"%G\t\t\t\t# dt [cm]\n", sim.dt);
		fprintf(file, "%hd\t%hd\t\t# No. of dt, da.\n\n",
			sim.nr, sim.na);
	
		fprintf(file,
			"%hd\t\t\t\t\t# Number of layers\n",
			sim.n_layers);
		fprintf(file,
			"#n\tmua\tmus\tg\td\t# One line for each layer\n");
		fprintf(file,
			"%G\t\t\t\t\t# n for medium above\n",
			sim.layers[0].n);
		for (i = 1; i <= sim.n_layers; i++)  {
			LayerStruct s;
			s = sim.layers[i];
			fprintf(file, "%G\t%G\t%G\t%G\t%G\t# layer %hd\n",
				s.n, s.mua, s.mutr, s.g, s.z_max - s.z_min, i);
		}
		fprintf(file, "%G\t\t\t\t\t# n for medium below\n\n",
			sim.layers[i].n);
	}
	__host__ void WriteRd_ra(FILE * file,
		short Nr,
		short Na,
		OutStruct Out_Parm)
	{
		short it, ia;

		fprintf(file,
			"%s\n%s\n%s\n%s\n%s\n%s\n",	/* flag. */
			"# Rd[theta][angle]. [1/(cm2sr)].",
			"# Rd[0][0], [0][1],..[0][na-1]",
			"# Rd[1][0], [1][1],..[1][na-1]",
			"# ...",
			"# Rd[nt-1][0], [nt-1][1],..[nt-1][na-1]",
			"Rd_ta");

		for (it = 0; it < Nr; it++)
		{
			for (ia = 0; ia < Na; ia++)
			{
				fprintf(file, "%12.4E,", Out_Parm.Rd_ra[it + Nr*ia]);
				if ((it*Na + ia + 1) % 9 == 0) fprintf(file, "\n");
			}
		}
		fprintf(file, "\n");
	}

	/***********************************************************
	*	1 number each line.
	****/

	__host__ void WriteRd_p(FILE * file,
		short Nr,
		short Na,
		OutStruct Out_Parm)
	{
		short it, ia;
	
		fprintf(file,
			"%s\n%s\n%s\n%s\n%s\n%s\n",	/* flag. */
			"# Rd[theta][angle]. [1/(cm2sr)].",
			"# Rd[1][0], [1][1],..[1][na-1]",
			"# ...",
			"# Rd[nt-1][0], [nt-1][1],..[nt-1][na-1]",
			"Rd_p");
	
		for (it = 0; it < Nr; it++)
		{
			for (ia = 0; ia < Na; ia++)
			{
				fprintf(file, "%12.4E,", Out_Parm.Rd_p[it + ia*Nr]);
				if ((it*Na + ia + 1) % 9 == 0) fprintf(file, "\n");
			}
		}
		fprintf(file, "photon number");
		fprintf(file, "%ld\n", Out_Parm.p1);
		fprintf(file, "\n");
	}

	/***********************************************************
	*	1 number each line.
	****/
	__host__ void WriteOPL(FILE * file,
		short nl,
		OutStruct Out_Parm)
	{
		short l;
	
		for (l = 1; l <= nl; l++)
		{
			fprintf(file, "The %d layer\n", l);
			fprintf(file, "%12.4E\n", Out_Parm.opl[l]);	/* ���ό��H���̏������� */
			fprintf(file, "\n");
		}
	}
	__host__ __device__ double *AllocMatrix(short nrl, short nrh, short ncl, int nch)
	{
		long i, j;
		double * m;

		m = (double *)malloc((unsigned)(nrh - nrl + 1)*(nch - ncl + 1) *sizeof(double*));
		//if (!m) nrerror("allocation failure 1 in matrix()");
		m -= nrl;

		for (i = nrl; i <= nrh; i++) {
		//m[i] = (double )malloc((unsigned)(nch - ncl + 1) *sizeof(double));
		//if (!m[i]) nrerror("allocation failure 2 in matrix()");
		m[i] -= ncl;
		}

		for (i = nrl; i <= nrh; i++)
			for (j = ncl; j <= nch; j++) m[i*(nrh - nrl + 1) + j] = 0.0;
		return m;
	}
}